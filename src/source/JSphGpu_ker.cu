#include "hip/hip_runtime.h"
//HEAD_DSPH
/*
 <DUALSPHYSICS>  Copyright (c) 2017 by Dr Jose M. Dominguez et al. (see http://dual.sphysics.org/index.php/developers/). 

 EPHYSLAB Environmental Physics Laboratory, Universidade de Vigo, Ourense, Spain.
 School of Mechanical, Aerospace and Civil Engineering, University of Manchester, Manchester, U.K.

 This file is part of DualSPHysics. 

 DualSPHysics is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser General Public License 
 as published by the Free Software Foundation; either version 2.1 of the License, or (at your option) any later version.
 
 DualSPHysics is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more details. 

 You should have received a copy of the GNU Lesser General Public License along with DualSPHysics. If not, see <http://www.gnu.org/licenses/>. 
*/

/// \file JSphGpu_ker.cu \brief Implements functions and CUDA kernels for the Particle Interaction and System Update.

#include "JSphGpu_ker.h"
#include "JBlockSizeAuto.h"
#include "JLog2.h"
#include <cfloat>
#include <hip/hip_math_constants.h>
//:#include "JDgKerPrint.h"
//:#include "JDgKerPrint_ker.h"

#pragma warning(disable : 4267) //Cancels "warning C4267: conversion from 'size_t' to 'int', possible loss of data"
#pragma warning(disable : 4244) //Cancels "warning C4244: conversion from 'unsigned __int64' to 'unsigned int', possible loss of data"
#pragma warning(disable : 4503) //Cancels "warning C4503: decorated name length exceeded, name was truncated"
#include <thrust/device_vector.h>
#include <thrust/sort.h>

__constant__ StCteInteraction CTE;


namespace cusph{
#include "FunctionsMath_ker.cu"

//==============================================================================
/// Checks error and ends execution.
/// Comprueba error y finaliza ejecucion.
//==============================================================================
#define CheckErrorCuda(text)  __CheckErrorCuda(text,__FILE__,__LINE__)
void __CheckErrorCuda(const char *text,const char *file,const int line){
  hipError_t err=hipGetLastError();
  if(hipSuccess!=err){
    char cad[2048]; 
    sprintf(cad,"%s (CUDA error: %s -> %s:%i).\n",text,hipGetErrorString(err),file,line); 
    throw std::string(cad);
  }
}

//==============================================================================
/// Returns size of gridsize according to parameters.
/// Devuelve tamaño de gridsize segun parametros.
//==============================================================================
dim3 GetGridSize(unsigned n,unsigned blocksize){
  dim3 sgrid;//=dim3(1,2,3);
  unsigned nb=unsigned(n+blocksize-1)/blocksize; //-Total number of blocks to execute.
  sgrid.x=(nb<=65535? nb: unsigned(sqrt(float(nb))));
  sgrid.y=(nb<=65535? 1: unsigned((nb+sgrid.x-1)/sgrid.x));
  sgrid.z=1;
  return(sgrid);
}

//==============================================================================
/// Reduction using maximum of float values in shared memory for a warp.
/// Reduccion mediante maximo de valores float en memoria shared para un warp.
//==============================================================================
template <unsigned blockSize> __device__ void KerReduMaxFloatWarp(volatile float* sdat,unsigned tid){
  if(blockSize>=64)sdat[tid]=max(sdat[tid],sdat[tid+32]);
  if(blockSize>=32)sdat[tid]=max(sdat[tid],sdat[tid+16]);
  if(blockSize>=16)sdat[tid]=max(sdat[tid],sdat[tid+8]);
  if(blockSize>=8)sdat[tid]=max(sdat[tid],sdat[tid+4]);
  if(blockSize>=4)sdat[tid]=max(sdat[tid],sdat[tid+2]);
  if(blockSize>=2)sdat[tid]=max(sdat[tid],sdat[tid+1]);
}

//==============================================================================
/// Accumulates the maximum of n values of array dat[], storing the result in 
/// the beginning of res[].(Many positions of res[] are used as blocks, 
/// storing the final result in res[0]).
///
/// Acumula el maximo de n valores del vector dat[], guardando el resultado al 
/// principio de res[] (Se usan tantas posiciones del res[] como bloques, 
/// quedando el resultado final en res[0]).
//==============================================================================
template <unsigned blockSize> __global__ void KerReduMaxFloat(unsigned n,unsigned ini,const float *dat,float *res){
  extern __shared__ float sdat[];
  unsigned tid=threadIdx.x;
  unsigned c=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
  sdat[tid]=(c<n? dat[c+ini]: -FLT_MAX);
  __syncthreads();
  if(blockSize>=512){ if(tid<256)sdat[tid]=max(sdat[tid],sdat[tid+256]);  __syncthreads(); }
  if(blockSize>=256){ if(tid<128)sdat[tid]=max(sdat[tid],sdat[tid+128]);  __syncthreads(); }
  if(blockSize>=128){ if(tid<64) sdat[tid]=max(sdat[tid],sdat[tid+64]);   __syncthreads(); }
  if(tid<32)KerReduMaxFloatWarp<blockSize>(sdat,tid);
  if(tid==0)res[blockIdx.y*gridDim.x + blockIdx.x]=sdat[0];
}

//==============================================================================
/// Returns the maximum of an array, using resu[] as auxiliar array.
/// Size of resu[] must be >= a (N/SPHBSIZE+1)+(N/(SPHBSIZE*SPHBSIZE)+SPHBSIZE)
///
/// Devuelve el maximo de un vector, usando resu[] como vector auxiliar. El tamaño
/// de resu[] debe ser >= a (N/SPHBSIZE+1)+(N/(SPHBSIZE*SPHBSIZE)+SPHBSIZE)
//==============================================================================
float ReduMaxFloat(unsigned ndata,unsigned inidata,float* data,float* resu){
  float resf;
  if (1) {  

	  unsigned n = ndata, ini = inidata;
	  unsigned smemSize = SPHBSIZE * sizeof(float);
	  dim3 sgrid = GetGridSize(n, SPHBSIZE);
	  unsigned n_blocks = sgrid.x*sgrid.y;
	  float *dat = data;
	  float *resu1 = resu, *resu2 = resu + n_blocks;
	  float *res = resu1;
	  while (n > 1) {
		  KerReduMaxFloat<SPHBSIZE> <<<sgrid, SPHBSIZE, smemSize >>> (n, ini, dat, res);
		  n = n_blocks; ini = 0;
		  sgrid = GetGridSize(n, SPHBSIZE);
		  n_blocks = sgrid.x*sgrid.y;
		  if (n > 1) {
			  dat = res; res = (dat == resu1 ? resu2 : resu1);
		  }
	  }

	  if (ndata > 1) {
		  hipMemcpy(&resf, res, sizeof(float), hipMemcpyDeviceToHost);
	  }
	  else {
			hipMemcpy(&resf, data, sizeof(float), hipMemcpyDeviceToHost);

	  }	  

  }

  //else{//-Using Thrust library is slower than ReduMasFloat() with ndata < 5M.
  //  thrust::device_ptr<float> dev_ptr(data);
  //  resf=thrust::reduce(dev_ptr,dev_ptr+ndata,-FLT_MAX,thrust::maximum<float>());
  //}

  return(resf);
}

//==============================================================================
/// Accumulates the sum of n values of array dat[], storing the result in 
/// the beginning of res[].(Many positions of res[] are used as blocks, 
/// storing the final result in res[0]).
///
/// Acumula la suma de n valores del vector dat[].w, guardando el resultado al 
/// principio de res[] (Se usan tantas posiciones del res[] como bloques, 
/// quedando el resultado final en res[0]).
//==============================================================================
template <unsigned blockSize> __global__ void KerReduMaxFloat_w(unsigned n,unsigned ini,const float4 *dat,float *res){
  extern __shared__ float sdat[];
  unsigned tid=threadIdx.x;
  unsigned c=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
  sdat[tid]=(c<n? dat[c+ini].w: -FLT_MAX);
  __syncthreads();
  if(blockSize>=512){ if(tid<256)sdat[tid]=max(sdat[tid],sdat[tid+256]);  __syncthreads(); }
  if(blockSize>=256){ if(tid<128)sdat[tid]=max(sdat[tid],sdat[tid+128]);  __syncthreads(); }
  if(blockSize>=128){ if(tid<64) sdat[tid]=max(sdat[tid],sdat[tid+64]);   __syncthreads(); }
  if(tid<32)KerReduMaxFloatWarp<blockSize>(sdat,tid);
  if(tid==0)res[blockIdx.y*gridDim.x + blockIdx.x]=sdat[0];
}

//==============================================================================
/// Returns the maximum of an array, using resu[] as auxiliar array.
/// Size of resu[] must be >= a (N/SPHBSIZE+1)+(N/(SPHBSIZE*SPHBSIZE)+SPHBSIZE).
///
/// Devuelve el maximo de la componente w de un vector float4, usando resu[] como 
/// vector auxiliar. El tamaño de resu[] debe ser >= a (N/SPHBSIZE+1)+(N/(SPHBSIZE*SPHBSIZE)+SPHBSIZE).
//==============================================================================
float ReduMaxFloat_w(unsigned ndata,unsigned inidata,float4* data,float* resu){
  unsigned n=ndata,ini=inidata;
  unsigned smemSize=SPHBSIZE*sizeof(float);
  dim3 sgrid=GetGridSize(n,SPHBSIZE);
  unsigned n_blocks=sgrid.x*sgrid.y;
  float *dat=NULL;
  float *resu1=resu,*resu2=resu+n_blocks;
  float *res=resu1;
  while(n>1){
    if(!dat)KerReduMaxFloat_w<SPHBSIZE><<<sgrid,SPHBSIZE,smemSize>>>(n,ini,data,res);
    else KerReduMaxFloat<SPHBSIZE><<<sgrid,SPHBSIZE,smemSize>>>(n,ini,dat,res);
    n=n_blocks; ini=0;
    sgrid=GetGridSize(n,SPHBSIZE);  
    n_blocks=sgrid.x*sgrid.y;
    if(n>1){
      dat=res; res=(dat==resu1? resu2: resu1); 
    }
  }
  float resf;
  if(ndata>1)hipMemcpy(&resf,res,sizeof(float),hipMemcpyDeviceToHost);
  else{
    float4 resf4;
    hipMemcpy(&resf4,data,sizeof(float4),hipMemcpyDeviceToHost);
    resf=resf4.w;
  }
  return(resf);
}

//==============================================================================
/// Stores constants for the GPU interaction.
/// Graba constantes para la interaccion a la GPU.
//==============================================================================
void CteInteractionUp(const StCteInteraction *cte){
  hipMemcpyToSymbol(HIP_SYMBOL(CTE),cte,sizeof(StCteInteraction));
}

//------------------------------------------------------------------------------
/// Initialises array with the indicated value.
/// Inicializa array con el valor indicado.
//------------------------------------------------------------------------------
__global__ void KerInitArray(unsigned n,float3 *v,float3 value)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n)v[p]=value;
}

//==============================================================================
/// Initialises array with the indicated value.
/// Inicializa array con el valor indicado.
//==============================================================================
void InitArray(unsigned n,float3 *v,tfloat3 value){
  if(n){
    dim3 sgrid=GetGridSize(n,SPHBSIZE);
    KerInitArray <<<sgrid,SPHBSIZE>>> (n,v,Float3(value));
  }
}

//------------------------------------------------------------------------------
/// Sets v[].y to zero.
/// Pone v[].y a cero.
//------------------------------------------------------------------------------
__global__ void KerResety(unsigned n,unsigned ini,float3 *v)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n)v[p+ini].y=0;
}

//==============================================================================
/// Sets v[].y to zero.
/// Pone v[].y a cero.
//==============================================================================
void Resety(unsigned n,unsigned ini,float3 *v){
  if(n){
    dim3 sgrid=GetGridSize(n,SPHBSIZE);
    KerResety <<<sgrid,SPHBSIZE>>> (n,ini,v);
  }
}

//------------------------------------------------------------------------------
/// Calculates module^2 of ace.
//------------------------------------------------------------------------------
__global__ void KerComputeAceMod(unsigned n,const float3 *ace,float *acemod)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const float3 r=ace[p];
    acemod[p]=r.x*r.x+r.y*r.y+r.z*r.z;
  }
}

//==============================================================================
/// Calculates module^2 of ace.
//==============================================================================
void ComputeAceMod(unsigned n,const float3 *ace,float *acemod){
  if(n){
    dim3 sgrid=GetGridSize(n,SPHBSIZE);
    KerComputeAceMod <<<sgrid,SPHBSIZE>>> (n,ace,acemod);
  }
}

//------------------------------------------------------------------------------
/// Calculates module^2 of ace, comprobando que la particula sea normal.
//------------------------------------------------------------------------------
__global__ void KerComputeAceMod(unsigned n,const typecode *code,const float3 *ace,float *acemod)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const float3 r=(CODE_IsNormal(code[p])? ace[p]: make_float3(0,0,0));
    acemod[p]=r.x*r.x+r.y*r.y+r.z*r.z;
  }
}

//==============================================================================
/// Calculates module^2 of ace, comprobando que la particula sea normal.
//==============================================================================
void ComputeAceMod(unsigned n,const typecode *code,const float3 *ace,float *acemod){
  if(n){
    dim3 sgrid=GetGridSize(n,SPHBSIZE);
    KerComputeAceMod <<<sgrid,SPHBSIZE>>> (n,code,ace,acemod);
  }
}


//##############################################################################
//# Other kernels...
//# Otros kernels...
//##############################################################################
//------------------------------------------------------------------------------
/// Calculates module^2 of vel.
//------------------------------------------------------------------------------
__global__ void KerComputeVelMod(unsigned n,const float4 *vel,float *velmod)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const float4 r=vel[p];
    velmod[p]=r.x*r.x+r.y*r.y+r.z*r.z;
	//printf("velmod : %f \n", velmod[p]);
  }
}

//==============================================================================
/// Calculates module^2 of vel.
//==============================================================================
void ComputeVelMod(unsigned n,const float4 *vel,float *velmod){
  if(n){
    dim3 sgrid=GetGridSize(n,SPHBSIZE);
    KerComputeVelMod <<<sgrid,SPHBSIZE>>> (n,vel,velmod);
  }
}


//##############################################################################
//# Kernels for preparing force computation with Pos-Single.
//# Kernels para preparar calculo de fuerzas con Pos-Single.
//##############################################################################
//------------------------------------------------------------------------------
/// Prepare variables for Pos-Single interaction.
/// Prepara variables para interaccion Pos-Single.
//------------------------------------------------------------------------------
__global__ void KerPreInteractionSingle(unsigned n,const double2 *posxy,const double *posz
  ,const float4 *velrhop,float4 *pospress,float cteb,float gamma)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Particle number.
  if(p<n){
    //Computes press in single or double precision,although the latter does not have any significant positive effect,
    //and like PosDouble if it is previously calculated and read the interaction can incur losses of
    //performance of 6% or 15% (GTX480 or k20c) so it is best calculated as always simple.
    //
    //Calcular press en simple o doble precision no parece tener ningun efecto positivo significativo,
    //y como para PosDouble si se calcula antes y se lee en la interaccion supondria una perdida de 
    //rendimiento del 6% o 15% (gtx480 o k20c) mejor se calcula en simple siempre.
    const float rrhop=velrhop[p].w;
    float press=cteb*(powf(rrhop*CTE.ovrhopzero,gamma)-1.0f);
	printf("KPISingle Press: %.8f\n", press);
    double2 rpos=posxy[p];
    pospress[p]=make_float4(float(rpos.x),float(rpos.y),float(posz[p]),press);
  }
}

//==============================================================================
/// Prepare variables for Pos-Single interaction.
/// Prepara variables para interaccion Pos-Single.
//==============================================================================
void PreInteractionSingle(unsigned np,const double2 *posxy,const double *posz
  ,const float4 *velrhop,float4 *pospress,float cteb,float ctegamma)
{
  if(np){
    dim3 sgrid=GetGridSize(np,SPHBSIZE);
    KerPreInteractionSingle <<<sgrid,SPHBSIZE>>> (np,posxy,posz,velrhop,pospress,cteb,ctegamma);
  }
}


//##############################################################################
//# Auxiliary kernels for the interaction.
//# Kernels auxiliares para interaccion.
//##############################################################################
//------------------------------------------------------------------------------
/// Returns position, vel, rhop and press of a particle.
/// Devuelve posicion, vel, rhop y press de particula.
//------------------------------------------------------------------------------
template<bool psingle> __device__ void KerGetParticleData(unsigned p1
  ,const double2 *posxy,const double *posz,const float4 *pospress,const float4 *velrhop
  ,float3 &velp1,float &rhopp1,double3 &posdp1,float3 &posp1,float &pressp1)
{
  float4 r=velrhop[p1];
  velp1=make_float3(r.x,r.y,r.z);
  rhopp1=r.w;
  if(psingle){
    float4 pxy=pospress[p1];
    posp1=make_float3(pxy.x,pxy.y,pxy.z);
    pressp1=pxy.w;
  }
  else{
    double2 pxy=posxy[p1];
    posdp1=make_double3(pxy.x,pxy.y,posz[p1]);
    pressp1=(CTE.cteb*(powf(rhopp1*CTE.ovrhopzero,CTE.gamma)-1.0f));
  }
}

//------------------------------------------------------------------------------
/// Returns postion and vel of a particle.
/// Devuelve posicion y vel de particula.
//------------------------------------------------------------------------------
template<bool psingle> __device__ void KerGetParticleData(unsigned p1
  ,const double2 *posxy,const double *posz,const float4 *pospress,const float4 *velrhop
  ,float3 &velp1,double3 &posdp1,float3 &posp1)
{
  float4 r=velrhop[p1];
  velp1=make_float3(r.x,r.y,r.z);
  if(psingle){
    float4 pxy=pospress[p1];
    posp1=make_float3(pxy.x,pxy.y,pxy.z);
  }
  else{
    double2 pxy=posxy[p1];
    posdp1=make_double3(pxy.x,pxy.y,posz[p1]);
  }
}

//------------------------------------------------------------------------------
/// Returns particle postion.
/// Devuelve posicion de particula.
//------------------------------------------------------------------------------
template<bool psingle> __device__ void KerGetParticleData(unsigned p1
  ,const double2 *posxy,const double *posz,const float4 *pospress
  ,double3 &posdp1,float3 &posp1)
{
  if(psingle){
    float4 pxy=pospress[p1];
    posp1=make_float3(pxy.x,pxy.y,pxy.z);
  }
  else{
    double2 pxy=posxy[p1];
    posdp1=make_double3(pxy.x,pxy.y,posz[p1]);
  }
}

//------------------------------------------------------------------------------
/// Returns drx, dry and drz between the particles.
/// Devuelve drx, dry y drz entre dos particulas.
//------------------------------------------------------------------------------
template<bool psingle> __device__ void KerGetParticlesDr(int p2
  ,const double2 *posxy,const double *posz,const float4 *pospress
  ,const double3 &posdp1,const float3 &posp1
  ,float &drx,float &dry,float &drz,float &pressp2)
{
  if(psingle){
    float4 posp2=pospress[p2];
    drx=posp1.x-posp2.x;
    dry=posp1.y-posp2.y;
    drz=posp1.z-posp2.z;
    pressp2=posp2.w;
  }
  else{
    double2 posp2=posxy[p2];
    drx=float(posdp1.x-posp2.x);
    dry=float(posdp1.y-posp2.y);
    drz=float(posdp1.z-posz[p2]);
    pressp2=0;
  }
}

//------------------------------------------------------------------------------
/// Returns drx, dry and drz between the particles.
/// Devuelve drx, dry y drz entre dos particulas.
//------------------------------------------------------------------------------
template<bool psingle> __device__ void KerGetParticlesDr(int p2
  ,const double2 *posxy,const double *posz,const float4 *pospress
  ,const double3 &posdp1,const float3 &posp1
  ,float &drx,float &dry,float &drz)
{
  if(psingle){
    float4 posp2=pospress[p2];
    drx=posp1.x-posp2.x;
    dry=posp1.y-posp2.y;
    drz=posp1.z-posp2.z;
  }
  else{
    double2 posp2=posxy[p2];
    drx=float(posdp1.x-posp2.x);
    dry=float(posdp1.y-posp2.y);
    drz=float(posdp1.z-posz[p2]);
  }
}

//------------------------------------------------------------------------------
/// Returns cell limits for the interaction.
/// Devuelve limites de celdas para interaccion.
//------------------------------------------------------------------------------
__device__ void KerGetInteractionCells(unsigned rcell
  ,int hdiv,const int4 &nc,const int3 &cellzero
  ,int &cxini,int &cxfin,int &yini,int &yfin,int &zini,int &zfin)
{
  //-Obtains interaction limits.
  const int cx=PC__Cellx(CTE.cellcode,rcell)-cellzero.x;
  const int cy=PC__Celly(CTE.cellcode,rcell)-cellzero.y;
  const int cz=PC__Cellz(CTE.cellcode,rcell)-cellzero.z;
  //-Code for hdiv 1 or 2 but not zero.
  //-Codigo para hdiv 1 o 2 pero no cero.
  cxini=cx-min(cx,hdiv);
  cxfin=cx+min(nc.x-cx-1,hdiv)+1;
  yini=cy-min(cy,hdiv);
  yfin=cy+min(nc.y-cy-1,hdiv)+1;
  zini=cz-min(cz,hdiv);
  zfin=cz+min(nc.z-cz-1,hdiv)+1;
}

//------------------------------------------------------------------------------
/// Returns cell limits for the interaction.
/// Devuelve limites de celdas para interaccion.
//------------------------------------------------------------------------------
__device__ void KerGetInteractionCells(double px,double py,double pz
  ,int hdiv,const int4 &nc,const int3 &cellzero
  ,int &cxini,int &cxfin,int &yini,int &yfin,int &zini,int &zfin)
{
  //-Obtains interaction limits.
  const int cx=int((px-CTE.domposminx)/CTE.scell)-cellzero.x;
  const int cy=int((py-CTE.domposminy)/CTE.scell)-cellzero.y;
  const int cz=int((pz-CTE.domposminz)/CTE.scell)-cellzero.z;
  //-Code for hdiv 1 or 2 but not zero.
  //-Codigo para hdiv 1 o 2 pero no cero.
  cxini=cx-min(cx,hdiv);
  cxfin=cx+min(nc.x-cx-1,hdiv)+1;
  yini=cy-min(cy,hdiv);
  yfin=cy+min(nc.y-cy-1,hdiv)+1;
  zini=cz-min(cz,hdiv);
  zfin=cz+min(nc.z-cz-1,hdiv)+1;
}

//------------------------------------------------------------------------------
/// Returns Wendland kernel values: frx, fry and frz.
/// Devuelve valores del kernel Wendland: frx, fry y frz.
//------------------------------------------------------------------------------
__device__ void KerGetKernelWendland(float rr2,float drx,float dry,float drz
  ,float &frx,float &fry,float &frz)
{
  const float rad=sqrt(rr2);
  const float qq=rad/CTE.h;
  //-Wendland kernel.
  const float wqq1=1.f-0.5f*qq;
  const float fac=CTE.bwen*qq*wqq1*wqq1*wqq1/rad;
  frx=fac*drx; fry=fac*dry; frz=fac*drz;
}

//------------------------------------------------------------------------------
/// Returns Gaussian kernel values: frx, fry and frz.
/// Devuelve valores del kernel Gaussian: frx, fry y frz.
//------------------------------------------------------------------------------
__device__ void KerGetKernelGaussian(float rr2,float drx,float dry,float drz
  ,float &frx,float &fry,float &frz)
{
  const float rad=sqrt(rr2);
  const float qq=rad/CTE.h;
  //-Gaussian kernel.
  const float qqexp=-4.0f*qq*qq;
  //const float wab=CTE.agau*expf(qqexp);
  const float fac=CTE.bgau*qq*expf(qqexp)/rad;
  frx=fac*drx; fry=fac*dry; frz=fac*drz;
}

//------------------------------------------------------------------------------
/// Return values of kernel Cubic without tensil correction, gradients: frx, fry and frz.
/// Devuelve valores de kernel Cubic sin correccion tensil, gradients: frx, fry y frz.
//------------------------------------------------------------------------------
__device__ void KerGetKernelCubic(float rr2,float drx,float dry,float drz
  ,float &frx,float &fry,float &frz)
{
  const float rad=sqrt(rr2);
  const float qq=rad/CTE.h;
  //-Cubic Spline kernel.
  float fac;
  if(rad>CTE.h){
    float wqq1=2.0f-qq;
    float wqq2=wqq1*wqq1;
    fac=CTE.cubic_c2*wqq2/rad;
  }
  else{
    float wqq2=qq*qq;
    fac=(CTE.cubic_c1*qq+CTE.cubic_d1*wqq2)/rad;
  }
  //-Gradients.
  frx=fac*drx; fry=fac*dry; frz=fac*drz;
}

//------------------------------------------------------------------------------
/// Return tensil correction for kernel Cubic.
/// Devuelve correccion tensil para kernel Cubic.
//------------------------------------------------------------------------------
__device__ float KerGetKernelCubicTensil(float rr2
  ,float rhopp1,float pressp1,float rhopp2,float pressp2)
{
  const float rad=sqrt(rr2);
  const float qq=rad/CTE.h;
  //-Cubic Spline kernel.
  float wab;
  if(rad>CTE.h){
    float wqq1=2.0f-qq;
    float wqq2=wqq1*wqq1;
    wab=CTE.cubic_a24*(wqq2*wqq1);
  }
  else{
    float wqq2=qq*qq;
    float wqq3=wqq2*qq;
    wab=CTE.cubic_a2*(1.0f-1.5f*wqq2+0.75f*wqq3);
  }
  //-Tensile correction.
  float fab=wab*CTE.cubic_odwdeltap;
  fab*=fab; fab*=fab; //fab=fab^4
  const float tensilp1=(pressp1/(rhopp1*rhopp1))*(pressp1>0? 0.01f: -0.2f);
  const float tensilp2=(pressp2/(rhopp2*rhopp2))*(pressp2>0? 0.01f: -0.2f);
  return(fab*(tensilp1+tensilp2));
}


//##############################################################################
//# Kernels for calculating forces (Pos-Double).
//# Kernels para calculo de fuerzas (Pos-Double).
//##############################################################################
//------------------------------------------------------------------------------
/// Interaction of a particle with a set of particles. Bound-Fluid/Float
/// Realiza la interaccion de una particula con un conjunto de ellas. Bound-Fluid/Float
//------------------------------------------------------------------------------
template<bool psingle,TpKernel tker,TpFtMode ftmode> __device__ void KerInteractionForcesBoundBox
  (unsigned p1,const unsigned &pini,const unsigned &pfin
  ,const float *ftomassp
  ,const double2 *posxy,const double *posz,const float4 *pospress,const float4 *velrhop,const typecode *code,const unsigned* idp
  ,float massf,double3 posdp1,float3 posp1,float3 velp1,float &arp1,float &visc)
{
  for(int p2=pini;p2<pfin;p2++){
    float drx,dry,drz;
    KerGetParticlesDr<psingle>(p2,posxy,posz,pospress,posdp1,posp1,drx,dry,drz);
    float rr2=drx*drx+dry*dry+drz*drz;
    if(rr2<=CTE.fourh2 && rr2>=ALMOSTZERO){
      //-Cubic Spline, Wendland or Gaussian kernel.
      float frx,fry,frz;
      if(tker==KERNEL_Wendland)KerGetKernelWendland(rr2,drx,dry,drz,frx,fry,frz);
      else if(tker==KERNEL_Gaussian)KerGetKernelGaussian(rr2,drx,dry,drz,frx,fry,frz);
      else if(tker==KERNEL_Cubic)KerGetKernelCubic(rr2,drx,dry,drz,frx,fry,frz);

      const float4 velrhop2=velrhop[p2];
      //-Obtains particle mass p2 if there are floating bodies.
      //-Obtiene masa de particula p2 en caso de existir floatings.
      float ftmassp2;    //-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massf si es fluid.
      bool compute=true; //-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      if(USE_FLOATING){
        const typecode cod=code[p2];
        bool ftp2=CODE_IsFloating(cod);
        ftmassp2=(ftp2? ftomassp[CODE_GetTypeValue(cod)]: massf);
        compute=!(USE_DEM && ftp2); //-Deactivated when DEM is used and is bound-float. | Se desactiva cuando se usa DEM y es bound-float.
      }

      if(compute){
        //-Density derivative.
        const float dvx=velp1.x-velrhop2.x, dvy=velp1.y-velrhop2.y, dvz=velp1.z-velrhop2.z;
        arp1+=(USE_FLOATING? ftmassp2: massf)*(dvx*frx+dvy*fry+dvz*frz);

        {//===== Viscosity ===== 
          const float dot=drx*dvx + dry*dvy + drz*dvz;
          const float dot_rr2=dot/(rr2+CTE.eta2);
          visc=max(dot_rr2,visc); 
        }
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Particle interaction. Bound-Fluid/Float
/// Realiza interaccion entre particulas. Bound-Fluid/Float
//------------------------------------------------------------------------------
template<bool psingle,TpKernel tker,TpFtMode ftmode> __global__ void KerInteractionForcesBound
  (unsigned n,int hdiv,int4 nc,const int2 *begincell,int3 cellzero,const unsigned *dcell
  ,const float *ftomassp
  ,const double2 *posxy,const double *posz,const float4 *pospress,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float *viscdt,float *ar)
{
  unsigned p1=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p1<n){
    float visc=0,arp1=0;

    //-Loads particle p1 data.
    double3 posdp1;
    float3 posp1,velp1;
    KerGetParticleData<psingle>(p1,posxy,posz,pospress,velrhop,velp1,posdp1,posp1);

    //-Obtains interaction limits.
    int cxini,cxfin,yini,yfin,zini,zfin;
    KerGetInteractionCells(dcell[p1],hdiv,nc,cellzero,cxini,cxfin,yini,yfin,zini,zfin);

    //-Boundary-Fluid interaction.
    for(int z=zini;z<zfin;z++){
      int zmod=(nc.w)*z+(nc.w*nc.z+1);//-Adds Nct + 1 which is the first cell fluid. | Le suma Nct+1 que es la primera celda de fluido.
      for(int y=yini;y<yfin;y++){
        int ymod=zmod+nc.x*y;
        unsigned pini,pfin=0;
        for(int x=cxini;x<cxfin;x++){
          int2 cbeg=begincell[x+ymod];
          if(cbeg.y){
            if(!pfin)pini=cbeg.x;
            pfin=cbeg.y;
          }
        }
        if(pfin)KerInteractionForcesBoundBox<psingle,tker,ftmode> (p1,pini,pfin,ftomassp,posxy,posz,pospress,velrhop,code,idp,CTE.massf,posdp1,posp1,velp1,arp1,visc);
      }
    }
    //-Stores results.
    if(arp1 || visc){
      ar[p1]+=arp1;
      if(visc>viscdt[p1])viscdt[p1]=visc;
    }
  }
}

//------------------------------------------------------------------------------
/// Interaction of a particle with a set of particles. (Fluid/Float-Fluid/Float/Bound)
/// Realiza la interaccion de una particula con un conjunto de ellas. (Fluid/Float-Fluid/Float/Bound)
//------------------------------------------------------------------------------
template<bool psingle,TpKernel tker,TpFtMode ftmode,bool lamsps,TpDeltaSph tdelta,bool shift> __device__ void KerInteractionForcesFluidBox
  (bool boundp2,unsigned p1,const unsigned &pini,const unsigned &pfin,float visco
  ,const float *ftomassp,const float2 *tauff
  ,const double2 *posxy,const double *posz,const float4 *pospress,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float massp2,float ftmassp1,bool ftp1
  ,double3 posdp1,float3 posp1,float3 velp1,float pressp1,float rhopp1
  ,const float2 &taup1_xx_xy,const float2 &taup1_xz_yy,const float2 &taup1_yz_zz
  ,float2 &grap1_xx_xy,float2 &grap1_xz_yy,float2 &grap1_yz_zz
  ,float3 &acep1,float &arp1,float &visc,float &deltap1
  ,TpShifting tshifting,float3 &shiftposp1,float &shiftdetectp1)
{
  for(int p2=pini;p2<pfin;p2++){
    float drx,dry,drz,pressp2;
    KerGetParticlesDr<psingle> (p2,posxy,posz,pospress,posdp1,posp1,drx,dry,drz,pressp2);
    float rr2=drx*drx+dry*dry+drz*drz;
    if(rr2<=CTE.fourh2 && rr2>=ALMOSTZERO){
      //-Cubic Spline, Wendland or Gaussian kernel.
      float frx,fry,frz;
      if(tker==KERNEL_Wendland)KerGetKernelWendland(rr2,drx,dry,drz,frx,fry,frz);
      else if(tker==KERNEL_Gaussian)KerGetKernelGaussian(rr2,drx,dry,drz,frx,fry,frz);
      else if(tker==KERNEL_Cubic)KerGetKernelCubic(rr2,drx,dry,drz,frx,fry,frz);

      //-Obtains mass of particle p2 if any floating bodies exist.
      //-Obtiene masa de particula p2 en caso de existir floatings.
      bool ftp2;         //-Indicates if it is floating. | Indica si es floating.
      float ftmassp2;    //-Contains mass of floating body or massf if fluid. | Contiene masa de particula floating o massp2 si es bound o fluid.
      bool compute=true; //-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      if(USE_FLOATING){
        const typecode cod=code[p2];
        ftp2=CODE_IsFloating(cod);
        ftmassp2=(ftp2? ftomassp[CODE_GetTypeValue(cod)]: massp2);
        #ifdef DELTA_HEAVYFLOATING
          if(ftp2 && ftmassp2<=(massp2*1.2f) && (tdelta==DELTA_Dynamic || tdelta==DELTA_DynamicExt))deltap1=FLT_MAX;
        #else
          if(ftp2 && (tdelta==DELTA_Dynamic || tdelta==DELTA_DynamicExt))deltap1=FLT_MAX;
        #endif
        if(ftp2 && shift && tshifting==SHIFT_NoBound)shiftposp1.x=FLT_MAX; //-Cancels shifting with floating bodies. | Con floatings anula shifting.
        compute=!(USE_DEM && ftp1 && (boundp2 || ftp2)); //-Deactivated when DEM is used and is float-float or float-bound. | Se desactiva cuando se usa DEM y es float-float o float-bound.
      }

      const float4 velrhop2=velrhop[p2];
      
      //===== Aceleration ===== 
      if(compute){
        if(!psingle)pressp2=(CTE.cteb*(powf(velrhop2.w*CTE.ovrhopzero,CTE.gamma)-1.0f));
        const float prs=(pressp1+pressp2)/(rhopp1*velrhop2.w) + (tker==KERNEL_Cubic? KerGetKernelCubicTensil(rr2,rhopp1,pressp1,velrhop2.w,pressp2): 0);
        const float p_vpm=-prs*(USE_FLOATING? ftmassp2*ftmassp1: massp2);
        acep1.x+=p_vpm*frx; acep1.y+=p_vpm*fry; acep1.z+=p_vpm*frz;
      }

      //-Density derivative.
      const float dvx=velp1.x-velrhop2.x, dvy=velp1.y-velrhop2.y, dvz=velp1.z-velrhop2.z;
      if(compute)arp1+=(USE_FLOATING? ftmassp2: massp2)*(dvx*frx+dvy*fry+dvz*frz);

      const float cbar=CTE.cs0;
      //-Density derivative (DeltaSPH Molteni).
      if((tdelta==DELTA_Dynamic || tdelta==DELTA_DynamicExt)){
        const float rhop1over2=rhopp1/velrhop2.w;
        const float visc_densi=CTE.delta2h*cbar*(rhop1over2-1.f)/(rr2+CTE.eta2);
        const float dot3=(drx*frx+dry*fry+drz*frz);
        const float delta=visc_densi*dot3*(USE_FLOATING? ftmassp2: massp2);
        if(USE_FLOATING)deltap1=(boundp2 || deltap1==FLT_MAX? FLT_MAX: deltap1+delta); //-Con floating bodies entre el fluido. //-For floating bodies within the fluid
        else deltap1=(boundp2? FLT_MAX: deltap1+delta);
      }

      //-Shifting correction.
      if(shift && shiftposp1.x!=FLT_MAX){
        const float massrhop=(USE_FLOATING? ftmassp2: massp2)/velrhop2.w;
        const bool noshift=(boundp2 && (tshifting==SHIFT_NoBound || (tshifting==SHIFT_NoFixed && CODE_IsFixed(code[p2]))));
        shiftposp1.x=(noshift? FLT_MAX: shiftposp1.x+massrhop*frx); //-Removes shifting for the boundaries. | Con boundary anula shifting.
        shiftposp1.y+=massrhop*fry;
        shiftposp1.z+=massrhop*frz;
        shiftdetectp1-=massrhop*(drx*frx+dry*fry+drz*frz);
      }

      //===== Viscosity ===== 
      if(compute){
        const float dot=drx*dvx + dry*dvy + drz*dvz;
        const float dot_rr2=dot/(rr2+CTE.eta2);
        visc=max(dot_rr2,visc);  //ViscDt=max(dot/(rr2+Eta2),ViscDt);
        if(!lamsps){//-Artificial viscosity.
          if(dot<0){
            const float amubar=CTE.h*dot_rr2;  //amubar=CTE.h*dot/(rr2+CTE.eta2);
            const float robar=(rhopp1+velrhop2.w)*0.5f;
            const float pi_visc=(-visco*cbar*amubar/robar)*(USE_FLOATING? ftmassp2*ftmassp1: massp2);
            acep1.x-=pi_visc*frx; acep1.y-=pi_visc*fry; acep1.z-=pi_visc*frz;
          }
        }
        else{//-Laminar+SPS viscosity.
          {//-Laminar contribution.
            const float robar2=(rhopp1+velrhop2.w);
            const float temp=4.f*visco/((rr2+CTE.eta2)*robar2);  //-Simplication of temp=2.0f*visco/((rr2+CTE.eta2)*robar); robar=(rhopp1+velrhop2.w)*0.5f;
            const float vtemp=(USE_FLOATING? ftmassp2: massp2)*temp*(drx*frx+dry*fry+drz*frz);  
            acep1.x+=vtemp*dvx; acep1.y+=vtemp*dvy; acep1.z+=vtemp*dvz;
          }
          //-SPS turbulence model.
          float2 taup2_xx_xy=taup1_xx_xy; //-taup1 is always zero when p1 is not fluid. | taup1 siempre es cero cuando p1 no es fluid.
          float2 taup2_xz_yy=taup1_xz_yy;
          float2 taup2_yz_zz=taup1_yz_zz;
          if(!boundp2 && (USE_NOFLOATING || !ftp2)){//-When p2 is fluid.
            float2 taup2=tauff[p2*3];     taup2_xx_xy.x+=taup2.x; taup2_xx_xy.y+=taup2.y;
                   taup2=tauff[p2*3+1];   taup2_xz_yy.x+=taup2.x; taup2_xz_yy.y+=taup2.y;
                   taup2=tauff[p2*3+2];   taup2_yz_zz.x+=taup2.x; taup2_yz_zz.y+=taup2.y;
          }
          acep1.x+=(USE_FLOATING? ftmassp2*ftmassp1: massp2)*(taup2_xx_xy.x*frx+taup2_xx_xy.y*fry+taup2_xz_yy.x*frz);
          acep1.y+=(USE_FLOATING? ftmassp2*ftmassp1: massp2)*(taup2_xx_xy.y*frx+taup2_xz_yy.y*fry+taup2_yz_zz.x*frz);
          acep1.z+=(USE_FLOATING? ftmassp2*ftmassp1: massp2)*(taup2_xz_yy.x*frx+taup2_yz_zz.x*fry+taup2_yz_zz.y*frz);
          //-Velocity gradients.
          if(USE_NOFLOATING || !ftp1){//-When p1 is fluid.
            const float volp2=-(USE_FLOATING? ftmassp2: massp2)/velrhop2.w;
            float dv=dvx*volp2; grap1_xx_xy.x+=dv*frx; grap1_xx_xy.y+=dv*fry; grap1_xz_yy.x+=dv*frz;
                  dv=dvy*volp2; grap1_xx_xy.y+=dv*frx; grap1_xz_yy.y+=dv*fry; grap1_yz_zz.x+=dv*frz;
                  dv=dvz*volp2; grap1_xz_yy.x+=dv*frx; grap1_yz_zz.x+=dv*fry; grap1_yz_zz.y+=dv*frz;
            // to compute tau terms we assume that gradvel.xy=gradvel.dudy+gradvel.dvdx, gradvel.xz=gradvel.dudz+gradvel.dwdx, gradvel.yz=gradvel.dvdz+gradvel.dwdy
            // so only 6 elements are needed instead of 3x3.
          }
        }
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Interaction between particles. Fluid/Float-Fluid/Float or Fluid/Float-Bound.
/// Includes artificial/laminar viscosity and normal/DEM floating bodies.
///
/// Realiza interaccion entre particulas. Fluid/Float-Fluid/Float or Fluid/Float-Bound
/// Incluye visco artificial/laminar y floatings normales/dem.
//------------------------------------------------------------------------------
template<bool psingle,TpKernel tker,TpFtMode ftmode,bool lamsps,TpDeltaSph tdelta,bool shift> __global__ void KerInteractionForcesFluid
  (unsigned n,unsigned pinit,int hdiv,int4 nc,unsigned cellfluid,float viscob,float viscof
  ,const int2 *begincell,int3 cellzero,const unsigned *dcell
  ,const float *ftomassp,const float2 *tauff,float2 *gradvelff
  ,const double2 *posxy,const double *posz,const float4 *pospress,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float *viscdt,float *ar,float3 *ace,float *delta
  ,TpShifting tshifting,float3 *shiftpos,float *shiftdetect)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    unsigned p1=p+pinit;      //-Number of particle.
    float visc=0,arp1=0,deltap1=0;
    float3 acep1=make_float3(0,0,0);

    //-Variables for Shifting.
    float3 shiftposp1;
    float shiftdetectp1;
    if(shift){
      shiftposp1=make_float3(0,0,0);
      shiftdetectp1=0;
    }

    //-Obtains data of particle p1 in case there are floating bodies.
    //-Obtiene datos de particula p1 en caso de existir floatings.
    bool ftp1;       //-Indicates if it is floating. | Indica si es floating.
    float ftmassp1;  //-Contains floating particle mass or 1.0f if it is fluid. | Contiene masa de particula floating o 1.0f si es fluid.
    if(USE_FLOATING){
      const typecode cod=code[p1];
      ftp1=CODE_IsFloating(cod);
      ftmassp1=(ftp1? ftomassp[CODE_GetTypeValue(cod)]: 1.f);
      if(ftp1 && (tdelta==DELTA_Dynamic || tdelta==DELTA_DynamicExt))deltap1=FLT_MAX;
      if(ftp1 && shift)shiftposp1.x=FLT_MAX; //-Shifting is not calculated for floating bodies. | Para floatings no se calcula shifting.
    }

    //-Obtains basic data of particle p1.
    double3 posdp1;
    float3 posp1,velp1;
    float rhopp1,pressp1;
    KerGetParticleData<psingle>(p1,posxy,posz,pospress,velrhop,velp1,rhopp1,posdp1,posp1,pressp1);
    
    //-Variables for Laminar+SPS.
    float2 taup1_xx_xy,taup1_xz_yy,taup1_yz_zz;
    if(lamsps){
      taup1_xx_xy=tauff[p1*3];
      taup1_xz_yy=tauff[p1*3+1];
      taup1_yz_zz=tauff[p1*3+2];
    }
    //-Variables for Laminar+SPS (computation).
    float2 grap1_xx_xy,grap1_xz_yy,grap1_yz_zz;
    if(lamsps){
      grap1_xx_xy=make_float2(0,0);
      grap1_xz_yy=make_float2(0,0);
      grap1_yz_zz=make_float2(0,0);
    }

    //-Obtains interaction limits.
    int cxini,cxfin,yini,yfin,zini,zfin;
    KerGetInteractionCells(dcell[p1],hdiv,nc,cellzero,cxini,cxfin,yini,yfin,zini,zfin);

    //-Interaction with fluids.
    for(int z=zini;z<zfin;z++){
      int zmod=(nc.w)*z+cellfluid; //-The sum showing where fluid cells start. | Le suma donde empiezan las celdas de fluido.
      for(int y=yini;y<yfin;y++){
        int ymod=zmod+nc.x*y;
        unsigned pini,pfin=0;
        for(int x=cxini;x<cxfin;x++){
          int2 cbeg=begincell[x+ymod];
          if(cbeg.y){
            if(!pfin)pini=cbeg.x;
            pfin=cbeg.y;
          }
        }
        if(pfin)KerInteractionForcesFluidBox<psingle,tker,ftmode,lamsps,tdelta,shift> (false,p1,pini,pfin,viscof,ftomassp,tauff,posxy,posz,pospress,velrhop,code,idp,CTE.massf,ftmassp1,ftp1,posdp1,posp1,velp1,pressp1,rhopp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy,grap1_xz_yy,grap1_yz_zz,acep1,arp1,visc,deltap1,tshifting,shiftposp1,shiftdetectp1);
      }
    }
    //-Interaction with boundaries.
    for(int z=zini;z<zfin;z++){
      int zmod=(nc.w)*z;
      for(int y=yini;y<yfin;y++){
        int ymod=zmod+nc.x*y;
        unsigned pini,pfin=0;
        for(int x=cxini;x<cxfin;x++){
          int2 cbeg=begincell[x+ymod];
          if(cbeg.y){
            if(!pfin)pini=cbeg.x;
            pfin=cbeg.y;
          }
        }
        if(pfin)KerInteractionForcesFluidBox<psingle,tker,ftmode,lamsps,tdelta,shift> (true ,p1,pini,pfin,viscob,ftomassp,tauff,posxy,posz,pospress,velrhop,code,idp,CTE.massb,ftmassp1,ftp1,posdp1,posp1,velp1,pressp1,rhopp1,taup1_xx_xy,taup1_xz_yy,taup1_yz_zz,grap1_xx_xy,grap1_xz_yy,grap1_yz_zz,acep1,arp1,visc,deltap1,tshifting,shiftposp1,shiftdetectp1);
      }
    }
    //-Stores results.
    if(shift||arp1||acep1.x||acep1.y||acep1.z||visc){
      if(tdelta==DELTA_Dynamic&&deltap1!=FLT_MAX)arp1+=deltap1;
      if(tdelta==DELTA_DynamicExt){
        float rdelta=delta[p1];
        delta[p1]=(rdelta==FLT_MAX||deltap1==FLT_MAX? FLT_MAX: rdelta+deltap1);
      }
      ar[p1]+=arp1;
      float3 r=ace[p1]; r.x+=acep1.x; r.y+=acep1.y; r.z+=acep1.z; ace[p1]=r;
      if(visc>viscdt[p1])viscdt[p1]=visc;
      if(lamsps){
        gradvelff[p1*3]=grap1_xx_xy;
        gradvelff[p1*3+1]=grap1_xz_yy;
        gradvelff[p1*3+2]=grap1_yz_zz;
      }
      if(shift){
        shiftpos[p1]=shiftposp1;
        if(shiftdetect)shiftdetect[p1]=shiftdetectp1;
      }
    }
  }
}


//------------------------------------------------------------------------------
/*template<bool psingle, TpKernel tker, TpFtMode ftmode, bool lamsps, TpDeltaSph tdelta, bool shift> __global__ void KerInteractionForcesSolid
(unsigned n, unsigned pinit, int hdiv, int4 nc, unsigned cellfluid, float viscob, float viscof
	, const int2 *begincell, int3 cellzero, const unsigned *dcell
	, const float *ftomassp, const float2 *tauff, float2 *gradvelff
	, const double2 *posxy, const double *posz, const float4 *pospress, const float4 *velrhop, const typecode *code, const unsigned *idp
	, const float pore, const float mass, const tsymatrix3f *tau
	, float *viscdt, float *ar, float3 *ace, float *delta
	, TpShifting tshifting, float3 *shiftpos, float *shiftdetect)
{
	unsigned p = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
	if (p<n) {
		unsigned p1 = p + pinit;      //-Number of particle.
		float visc = 0, arp1 = 0, deltap1 = 0;
		float3 acep1 = make_float3(0, 0, 0);

		//-Variables for Shifting.
		float3 shiftposp1;
		float shiftdetectp1;
		if (shift) {
			shiftposp1 = make_float3(0, 0, 0);
			shiftdetectp1 = 0;
		}

		//-Obtains data of particle p1 in case there are floating bodies.
		//-Obtiene datos de particula p1 en caso de existir floatings.
		bool ftp1;       //-Indicates if it is floating. | Indica si es floating.
		float ftmassp1;  //-Contains floating particle mass or 1.0f if it is fluid. | Contiene masa de particula floating o 1.0f si es fluid.
		if (USE_FLOATING) {
			const typecode cod = code[p1];
			ftp1 = CODE_IsFloating(cod);
			ftmassp1 = (ftp1 ? ftomassp[CODE_GetTypeValue(cod)] : 1.f);
			if (ftp1 && (tdelta == DELTA_Dynamic || tdelta == DELTA_DynamicExt))deltap1 = FLT_MAX;
			if (ftp1 && shift)shiftposp1.x = FLT_MAX; //-Shifting is not calculated for floating bodies. | Para floatings no se calcula shifting.
		}

		//-Obtains basic data of particle p1.
		double3 posdp1;
		float3 posp1, velp1;
		float rhopp1, pressp1;
		KerGetParticleData<psingle>(p1, posxy, posz, pospress, velrhop, velp1, rhopp1, posdp1, posp1, pressp1);

		//-Variables for Laminar+SPS.
		float2 taup1_xx_xy, taup1_xz_yy, taup1_yz_zz;
		if (lamsps) {
			taup1_xx_xy = tauff[p1 * 3];
			taup1_xz_yy = tauff[p1 * 3 + 1];
			taup1_yz_zz = tauff[p1 * 3 + 2];
		}
		//-Variables for Laminar+SPS (computation).
		float2 grap1_xx_xy, grap1_xz_yy, grap1_yz_zz;
		if (lamsps) {
			grap1_xx_xy = make_float2(0, 0);
			grap1_xz_yy = make_float2(0, 0);
			grap1_yz_zz = make_float2(0, 0);
		}

		//-Obtains interaction limits.
		int cxini, cxfin, yini, yfin, zini, zfin;
		KerGetInteractionCells(dcell[p1], hdiv, nc, cellzero, cxini, cxfin, yini, yfin, zini, zfin);

		//-Interaction with fluids.
		for (int z = zini; z<zfin; z++) {
			int zmod = (nc.w)*z + cellfluid; //-The sum showing where fluid cells start. | Le suma donde empiezan las celdas de fluido.
			for (int y = yini; y<yfin; y++) {
				int ymod = zmod + nc.x*y;
				unsigned pini, pfin = 0;
				for (int x = cxini; x<cxfin; x++) {
					int2 cbeg = begincell[x + ymod];
					if (cbeg.y) {
						if (!pfin)pini = cbeg.x;
						pfin = cbeg.y;
					}
				}
				if (pfin)KerInteractionForcesFluidBox<psingle, tker, ftmode, lamsps, tdelta, shift>(false, p1, pini, pfin, viscof, ftomassp, tauff, posxy, posz, pospress, velrhop, code, idp, CTE.massf, ftmassp1, ftp1, posdp1, posp1, velp1, pressp1, rhopp1, taup1_xx_xy, taup1_xz_yy, taup1_yz_zz, grap1_xx_xy, grap1_xz_yy, grap1_yz_zz, acep1, arp1, visc, deltap1, tshifting, shiftposp1, shiftdetectp1);
			}
		}
		//-Interaction with boundaries.
		for (int z = zini; z<zfin; z++) {
			int zmod = (nc.w)*z;
			for (int y = yini; y<yfin; y++) {
				int ymod = zmod + nc.x*y;
				unsigned pini, pfin = 0;
				for (int x = cxini; x<cxfin; x++) {
					int2 cbeg = begincell[x + ymod];
					if (cbeg.y) {
						if (!pfin)pini = cbeg.x;
						pfin = cbeg.y;
					}
				}
				if (pfin)KerInteractionForcesFluidBox<psingle, tker, ftmode, lamsps, tdelta, shift>(true, p1, pini, pfin, viscob, ftomassp, tauff, posxy, posz, pospress, velrhop, code, idp, CTE.massb, ftmassp1, ftp1, posdp1, posp1, velp1, pressp1, rhopp1, taup1_xx_xy, taup1_xz_yy, taup1_yz_zz, grap1_xx_xy, grap1_xz_yy, grap1_yz_zz, acep1, arp1, visc, deltap1, tshifting, shiftposp1, shiftdetectp1);
			}
		}
		//-Stores results.
		if (shift || arp1 || acep1.x || acep1.y || acep1.z || visc) {
			if (tdelta == DELTA_Dynamic && deltap1 != FLT_MAX)arp1 += deltap1;
			if (tdelta == DELTA_DynamicExt) {
				float rdelta = delta[p1];
				delta[p1] = (rdelta == FLT_MAX || deltap1 == FLT_MAX ? FLT_MAX : rdelta + deltap1);
			}
			ar[p1] += arp1;
			float3 r = ace[p1]; r.x += acep1.x; r.y += acep1.y; r.z += acep1.z; ace[p1] = r;
			if (visc>viscdt[p1])viscdt[p1] = visc;
			if (lamsps) {
				gradvelff[p1 * 3] = grap1_xx_xy;
				gradvelff[p1 * 3 + 1] = grap1_xz_yy;
				gradvelff[p1 * 3 + 2] = grap1_yz_zz;
			}
			if (shift) {
				shiftpos[p1] = shiftposp1;
				if (shiftdetect)shiftdetect[p1] = shiftdetectp1;
			}
		}
	}
}


, const float pore, const float mass, const tsymatrix3f *tau
, tsymatrix3f *taudot, tsymatrix3f *straindot, tsymatrix3f *spin*/

#ifndef DISABLE_BSMODES


//==============================================================================
/// Collects kernel information.
//==============================================================================
template<bool psingle,TpKernel tker,TpFtMode ftmode,bool lamsps,TpDeltaSph tdelta,bool shift> void Interaction_ForcesT_KerInfo
  (StKerInfo *kerinfo)
{
#if CUDART_VERSION >= 6050
  {
    typedef void (*fun_ptr)(unsigned,unsigned,int,int4,unsigned,float,float,const int2*,int3,const unsigned*,const float*,const float2*,float2*,const double2*,const double*,const float4*,const float4*,const typecode*,const unsigned*,float*,float*,float3*,float*,TpShifting,float3*,float*);
    fun_ptr ptr=&KerInteractionForcesFluid<psingle,tker,ftmode,lamsps,tdelta,shift>;
    int qblocksize=0,mingridsize=0;
    hipOccupancyMaxPotentialBlockSize(&mingridsize,&qblocksize,(void*)ptr,0,0);
    struct hipFuncAttributes attr;
    hipFuncGetAttributes(&attr,reinterpret_cast<const void*>((void*))ptr);
    kerinfo->forcesfluid_bs=qblocksize;
    kerinfo->forcesfluid_rg=attr.numRegs;
    kerinfo->forcesfluid_bsmax=attr.maxThreadsPerBlock;
    //printf(">> KerInteractionForcesFluid  blocksize:%u (%u)\n",qblocksize,0);
  }
  {
    typedef void (*fun_ptr)(unsigned,int,int4,const int2*,int3,const unsigned*,const float*,const double2*,const double*,const float4*,const float4*,const typecode*,const unsigned*,float*,float*);
    fun_ptr ptr=&KerInteractionForcesBound<psingle,tker,ftmode>;
    int qblocksize=0,mingridsize=0;
    hipOccupancyMaxPotentialBlockSize(&mingridsize,&qblocksize,(void*)ptr,0,0);
    struct hipFuncAttributes attr;
    hipFuncGetAttributes(&attr,reinterpret_cast<const void*>((void*))ptr);
    kerinfo->forcesbound_bs=qblocksize;
    kerinfo->forcesbound_rg=attr.numRegs;
    kerinfo->forcesbound_bsmax=attr.maxThreadsPerBlock;
    //printf(">> KerInteractionForcesBound  blocksize:%u (%u)\n",qblocksize,0);
  }
  CheckErrorCuda("Error collecting kernel information.");
#endif
}

//==============================================================================
/// Interaction for the force computation.
/// Interaccion para el calculo de fuerzas.
//==============================================================================
template<bool psingle,TpKernel tker,TpFtMode ftmode,bool lamsps,TpDeltaSph tdelta,bool shift> void Interaction_ForcesT_BsAuto
  (TpCellMode cellmode,float viscob,float viscof,unsigned bsbound,unsigned bsfluid
  ,unsigned np,unsigned npb,unsigned npbok,tuint3 ncells
  ,const int2 *begincell,tuint3 cellmin,const unsigned *dcell
  ,const double2 *posxy,const double *posz,const float4 *pospress
  ,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,const float *ftomassp,const tsymatrix3f *tau,tsymatrix3f *gradvel
  ,float *viscdt,float* ar,float3 *ace,float *delta
  ,TpShifting tshifting,float3 *shiftpos,float *shiftdetect
  ,bool simulate2d,JBlockSizeAuto *bsauto)
{
  if(1){
    //-Executes particle interactions.
    const unsigned npf=np-npb;
    const int hdiv=(cellmode==CELLMODE_H? 2: 1);
    const int4 nc=make_int4(int(ncells.x),int(ncells.y),int(ncells.z),int(ncells.x*ncells.y));
    const unsigned cellfluid=nc.w*nc.z+1;
    const int3 cellzero=make_int3(cellmin.x,cellmin.y,cellmin.z);
    //-Interaction Fluid-Fluid & Fluid-Bound.
    if(npf){
      JBlockSizeAutoKer* ker=bsauto->GetKernel(0);
      for(int ct=0;ct<ker->BsNum;ct++)if(ker->IsActive(ct)){
        hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start,0);
        unsigned bsize=ker->GetBs(ct);
        dim3 sgridf=GetGridSize(npf,bsize);
        KerInteractionForcesFluid<psingle,tker,ftmode,lamsps,tdelta,shift> <<<sgridf,bsize>>> (npf,npb,hdiv,nc,cellfluid,viscob,viscof,begincell,cellzero,dcell,ftomassp,(const float2*)tau,(float2*)gradvel,posxy,posz,pospress,velrhop,code,idp,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        float time;
        hipEventElapsedTime(&time,start,stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipError_t err=hipGetLastError();
        if(err!=hipSuccess)time=FLT_MAX;
        ker->SetTime(ct,time);
      }
    }
    //-Interaction Boundary-Fluid.
    if(npbok){
      JBlockSizeAutoKer* ker=bsauto->GetKernel(1);
      for(int ct=0;ct<ker->BsNum;ct++)if(ker->IsActive(ct)){
        hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start,0);
        unsigned bsize=ker->GetBs(ct);
        dim3 sgridb=GetGridSize(npbok,bsize);
        KerInteractionForcesBound<psingle,tker,ftmode> <<<sgridb,bsize>>> (npbok,hdiv,nc,begincell,cellzero,dcell,ftomassp,posxy,posz,pospress,velrhop,code,idp,viscdt,ar);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        float time;
        hipEventElapsedTime(&time,start,stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipError_t err=hipGetLastError();
        if(err!=hipSuccess)time=FLT_MAX;
        ker->SetTime(ct,time);
      }
    }
  }
}
#endif

//==============================================================================
/// Interaction for the force computation.
/// Interaccion para el calculo de fuerzas.
//==============================================================================
template<bool psingle,TpKernel tker,TpFtMode ftmode,bool lamsps,TpDeltaSph tdelta,bool shift> void Interaction_ForcesT
  (TpCellMode cellmode,float viscob,float viscof,unsigned bsbound,unsigned bsfluid
  ,unsigned np,unsigned npb,unsigned npbok,tuint3 ncells
  ,const int2 *begincell,tuint3 cellmin,const unsigned *dcell
  ,const double2 *posxy,const double *posz,const float4 *pospress
  ,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,const float *ftomassp,const tsymatrix3f *tau,tsymatrix3f *gradvel
  ,float *viscdt,float* ar,float3 *ace,float *delta
  ,TpShifting tshifting,float3 *shiftpos,float *shiftdetect
  ,bool simulate2d,StKerInfo *kerinfo,JBlockSizeAuto *bsauto)
{
  //-Collects kernel information.
#ifndef DISABLE_BSMODES
  if(kerinfo)Interaction_ForcesT_KerInfo<psingle,tker,ftmode,lamsps,tdelta,shift>(kerinfo);
  else if(bsauto)Interaction_ForcesT_BsAuto<psingle,tker,ftmode,lamsps,tdelta,shift>(cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,bsauto);
  else{
#endif
    //-Executes particle interactions.
    const unsigned npf=np-npb;
    const int hdiv=(cellmode==CELLMODE_H? 2: 1);
    const int4 nc=make_int4(int(ncells.x),int(ncells.y),int(ncells.z),int(ncells.x*ncells.y));
    const unsigned cellfluid=nc.w*nc.z+1;
    const int3 cellzero=make_int3(cellmin.x,cellmin.y,cellmin.z);
    //-Interaction Fluid-Fluid & Fluid-Bound.
    if(npf){
      dim3 sgridf=GetGridSize(npf,bsfluid);
      //printf("---->bsfluid:%u   ",bsfluid);
      KerInteractionForcesFluid<psingle,tker,ftmode,lamsps,tdelta,shift> <<<sgridf,bsfluid>>> (npf,npb,hdiv,nc,cellfluid,viscob,viscof,begincell,cellzero,dcell,ftomassp,(const float2*)tau,(float2*)gradvel,posxy,posz,pospress,velrhop,code,idp,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect);
    }
    //-Interaction Boundary-Fluid.
    if(npbok){
      dim3 sgridb=GetGridSize(npbok,bsbound);
      //printf("bsbound:%u\n",bsbound);
      KerInteractionForcesBound<psingle,tker,ftmode> <<<sgridb,bsbound>>> (npbok,hdiv,nc,begincell,cellzero,dcell,ftomassp,posxy,posz,pospress,velrhop,code,idp,viscdt,ar);
    }
#ifndef DISABLE_BSMODES
  }
#endif
}


//==============================================================================
// Matthias - Surcharge
template<bool psingle, TpKernel tker, TpFtMode ftmode, bool lamsps, TpDeltaSph tdelta, bool shift> void Interaction_ForcesT
(TpCellMode cellmode, float viscob, float viscof, unsigned bsbound, unsigned bsfluid
	, unsigned np, unsigned npb, unsigned npbok, tuint3 ncells
	, const int2 *begincell, tuint3 cellmin, const unsigned *dcell
	, const double2 *posxy, const double *posz, const float4 *pospress
	, const float4 *velrhop, const typecode *code, const unsigned *idp
	, const float *pore, const float *mass, const tsymatrix3f *tau
	, tsymatrix3f *taudot, tsymatrix3f *straindot, tsymatrix3f *spin
	, float *viscdt, float* ar, float3 *ace, float *delta
	, TpShifting tshifting, float3 *shiftpos, float *shiftdetect
	, bool simulate2d, StKerInfo *kerinfo, JBlockSizeAuto *bsauto)
{
	//-Collects kernel information.
#ifndef DISABLE_BSMODES
	//if (kerinfo)Interaction_ForcesT_KerInfo<psingle, tker, ftmode, lamsps, tdelta, shift>(kerinfo);
	//else if (bsauto)Interaction_ForcesT_BsAuto<psingle, tker, ftmode, lamsps, tdelta, shift>(cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, ftomassp, tau, gradvel, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, bsauto);
	//else {
#endif
		//-Executes particle interactions.
		const unsigned npf = np - npb;
		//const int hdiv = (cellmode == CELLMODE_H ? 2 : 1);
		const int4 nc = make_int4(int(ncells.x), int(ncells.y), int(ncells.z), int(ncells.x*ncells.y));
		//const unsigned cellfluid = nc.w*nc.z + 1;
		const int3 cellzero = make_int3(cellmin.x, cellmin.y, cellmin.z);
		//-Interaction Fluid-Fluid & Fluid-Bound.
		if (npf) {
			dim3 sgridf = GetGridSize(npf, bsfluid);
			//printf("---->bsfluid:%u   ",bsfluid);
			//KerInteractionForcesFluid<psingle, tker, ftmode, lamsps, tdelta, shift> << <sgridf, bsfluid >> > (npf, npb, hdiv, nc, cellfluid, viscob, viscof, begincell, cellzero, dcell, ftomassp, (const float2*)tau, (float2*)gradvel, posxy, posz, pospress, velrhop, code, idp, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect);
		}
		//-Interaction Boundary-Fluid.
		if (npbok) {
			dim3 sgridb = GetGridSize(npbok, bsbound);
			//printf("bsbound:%u\n",bsbound);
			//KerInteractionForcesBound<psingle, tker, ftmode> << <sgridb, bsbound >> > (npbok, hdiv, nc, begincell, cellzero, dcell, ftomassp, posxy, posz, pospress, velrhop, code, idp, viscdt, ar);
		}
#ifndef DISABLE_BSMODES
	//}
#endif
}

//==============================================================================
template<bool psingle,TpKernel tker,TpFtMode ftmode,bool lamsps> void Interaction_Forces_t2(TpDeltaSph tdelta,TpCellMode cellmode
  ,float viscob,float viscof,unsigned bsbound,unsigned bsfluid
  ,unsigned np,unsigned npb,unsigned npbok,tuint3 ncells
  ,const int2 *begincell,tuint3 cellmin,const unsigned *dcell
  ,const double2 *posxy,const double *posz,const float4 *pospress
  ,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,const float *ftomassp,const tsymatrix3f *tau,tsymatrix3f *gradvel
  ,float *viscdt,float* ar,float3 *ace,float *delta
  ,TpShifting tshifting,float3 *shiftpos,float *shiftdetect
  ,bool simulate2d,StKerInfo *kerinfo,JBlockSizeAuto *bsauto)
{
  if(tshifting){                const bool shift=true;
    if(tdelta==DELTA_None)      Interaction_ForcesT<psingle,tker,ftmode,lamsps,DELTA_None,shift>       (cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
    if(tdelta==DELTA_Dynamic)   Interaction_ForcesT<psingle,tker,ftmode,lamsps,DELTA_Dynamic,shift>    (cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
    if(tdelta==DELTA_DynamicExt)Interaction_ForcesT<psingle,tker,ftmode,lamsps,DELTA_DynamicExt,shift> (cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
  }
  else{                         const bool shift=false;
    if(tdelta==DELTA_None)      Interaction_ForcesT<psingle,tker,ftmode,lamsps,DELTA_None,shift>       (cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
    if(tdelta==DELTA_Dynamic)   Interaction_ForcesT<psingle,tker,ftmode,lamsps,DELTA_Dynamic,shift>    (cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
    if(tdelta==DELTA_DynamicExt)Interaction_ForcesT<psingle,tker,ftmode,lamsps,DELTA_DynamicExt,shift> (cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
  }
}

//==============================================================================
// Matthias surcharge
template<bool psingle, TpKernel tker, TpFtMode ftmode, bool lamsps> void Interaction_Forces_t2(TpDeltaSph tdelta, TpCellMode cellmode
	, float viscob, float viscof, unsigned bsbound, unsigned bsfluid
	, unsigned np, unsigned npb, unsigned npbok, tuint3 ncells
	, const int2 *begincell, tuint3 cellmin, const unsigned *dcell
	, const double2 *posxy, const double *posz, const float4 *pospress
	, const float4 *velrhop, const typecode *code, const unsigned *idp
	, const float *pore, const float *mass, const tsymatrix3f *tau
	, tsymatrix3f *taudot, tsymatrix3f *straindot, tsymatrix3f *spin
	, float *viscdt, float* ar, float3 *ace, float *delta
	, TpShifting tshifting, float3 *shiftpos, float *shiftdetect
	, bool simulate2d, StKerInfo *kerinfo, JBlockSizeAuto *bsauto)
{
	if (tshifting) {
		const bool shift = true;
		if (tdelta == DELTA_None)      Interaction_ForcesT<psingle, tker, ftmode, lamsps, DELTA_None, shift>(cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
		if (tdelta == DELTA_Dynamic)   Interaction_ForcesT<psingle, tker, ftmode, lamsps, DELTA_Dynamic, shift>(cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
		if (tdelta == DELTA_DynamicExt)Interaction_ForcesT<psingle, tker, ftmode, lamsps, DELTA_DynamicExt, shift>(cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
	}
	else {
		const bool shift = false;
		if (tdelta == DELTA_None)      Interaction_ForcesT<psingle, tker, ftmode, lamsps, DELTA_None, shift>(cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
		if (tdelta == DELTA_Dynamic)   Interaction_ForcesT<psingle, tker, ftmode, lamsps, DELTA_Dynamic, shift>(cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
		if (tdelta == DELTA_DynamicExt)Interaction_ForcesT<psingle, tker, ftmode, lamsps, DELTA_DynamicExt, shift>(cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
	}
}

//==============================================================================
template<bool psingle,TpKernel tker,TpFtMode ftmode> void Interaction_Forces_t1(bool lamsps,TpDeltaSph tdelta,TpCellMode cellmode
  ,float viscob,float viscof,unsigned bsbound,unsigned bsfluid
  ,unsigned np,unsigned npb,unsigned npbok,tuint3 ncells
  ,const int2 *begincell,tuint3 cellmin,const unsigned *dcell
  ,const double2 *posxy,const double *posz,const float4 *pospress
  ,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,const float *ftomassp,const tsymatrix3f *tau,tsymatrix3f *gradvel
  ,float *viscdt,float* ar,float3 *ace,float *delta
  ,TpShifting tshifting,float3 *shiftpos,float *shiftdetect
  ,bool simulate2d,StKerInfo *kerinfo,JBlockSizeAuto *bsauto)
{
  if(lamsps)Interaction_Forces_t2<psingle,tker,ftmode,true>  (tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
  else      Interaction_Forces_t2<psingle,tker,ftmode,false> (tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
}

//==============================================================================
// Matthias surcharge
template<bool psingle, TpKernel tker, TpFtMode ftmode> void Interaction_Forces_t1(bool lamsps, TpDeltaSph tdelta, TpCellMode cellmode
	, float viscob, float viscof, unsigned bsbound, unsigned bsfluid
	, unsigned np, unsigned npb, unsigned npbok, tuint3 ncells
	, const int2 *begincell, tuint3 cellmin, const unsigned *dcell
	, const double2 *posxy, const double *posz, const float4 *pospress
	, const float4 *velrhop, const typecode *code, const unsigned *idp
	, const float *pore, const float *mass, const tsymatrix3f *tau
	, tsymatrix3f *taudot, tsymatrix3f *straindot, tsymatrix3f *spin
	, float *viscdt, float* ar, float3 *ace, float *delta
	, TpShifting tshifting, float3 *shiftpos, float *shiftdetect
	, bool simulate2d, StKerInfo *kerinfo, JBlockSizeAuto *bsauto)
{
	if (lamsps)Interaction_Forces_t2<psingle, tker, ftmode, true>(tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
	else      Interaction_Forces_t2<psingle, tker, ftmode, false>(tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
}


//==============================================================================
/// Original Interaction_Forces
void Interaction_Forces(bool psingle,TpKernel tkernel,bool floating,bool usedem,bool lamsps
  ,TpDeltaSph tdelta,TpCellMode cellmode
  ,float viscob,float viscof,unsigned bsbound,unsigned bsfluid
  ,unsigned np,unsigned npb,unsigned npbok,tuint3 ncells
  ,const int2 *begincell,tuint3 cellmin,const unsigned *dcell
  ,const double2 *posxy,const double *posz,const float4 *pospress
  ,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,const float *ftomassp,const tsymatrix3f *tau,tsymatrix3f *gradvel
  ,float *viscdt,float* ar,float3 *ace,float *delta
  ,TpShifting tshifting,float3 *shiftpos,float *shiftdetect
  ,bool simulate2d,StKerInfo *kerinfo,JBlockSizeAuto *bsauto)
{
  if(tkernel==KERNEL_Wendland){    const TpKernel tker=KERNEL_Wendland;
    if(psingle){      const bool psingle=true;
      if(!floating)   Interaction_Forces_t1<psingle,tker,FTMODE_None> (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
      else if(!usedem)Interaction_Forces_t1<psingle,tker,FTMODE_Sph>  (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
      else            Interaction_Forces_t1<psingle,tker,FTMODE_Dem>  (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
    }else{            const bool psingle=false;
      if(!floating)   Interaction_Forces_t1<psingle,tker,FTMODE_None> (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
      else if(!usedem)Interaction_Forces_t1<psingle,tker,FTMODE_Sph>  (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
      else            Interaction_Forces_t1<psingle,tker,FTMODE_Dem>  (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
    }
  }else if(tkernel==KERNEL_Gaussian){ const TpKernel tker=KERNEL_Gaussian;
    if(psingle){      const bool psingle=true;
      if(!floating)   Interaction_Forces_t1<psingle,tker,FTMODE_None> (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
      else if(!usedem)Interaction_Forces_t1<psingle,tker,FTMODE_Sph>  (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
      else            Interaction_Forces_t1<psingle,tker,FTMODE_Dem>  (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
    }else{            const bool psingle=false;
      if(!floating)   Interaction_Forces_t1<psingle,tker,FTMODE_None> (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
      else if(!usedem)Interaction_Forces_t1<psingle,tker,FTMODE_Sph>  (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
      else            Interaction_Forces_t1<psingle,tker,FTMODE_Dem>  (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
    }
  }else if(tkernel==KERNEL_Cubic){ const TpKernel tker=KERNEL_Cubic;
    if(psingle){      const bool psingle=true;
      if(!floating)   Interaction_Forces_t1<psingle,tker,FTMODE_None> (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
      else if(!usedem)Interaction_Forces_t1<psingle,tker,FTMODE_Sph>  (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
      else            Interaction_Forces_t1<psingle,tker,FTMODE_Dem>  (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
    }else{            const bool psingle=false;
      if(!floating)   Interaction_Forces_t1<psingle,tker,FTMODE_None> (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
      else if(!usedem)Interaction_Forces_t1<psingle,tker,FTMODE_Sph>  (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
      else            Interaction_Forces_t1<psingle,tker,FTMODE_Dem>  (lamsps,tdelta,cellmode,viscob,viscof,bsbound,bsfluid,np,npb,npbok,ncells,begincell,cellmin,dcell,posxy,posz,pospress,velrhop,code,idp,ftomassp,tau,gradvel,viscdt,ar,ace,delta,tshifting,shiftpos,shiftdetect,simulate2d,kerinfo,bsauto);
    }
  }
}


//==============================================================================
/// Interaction_Forces with Solid and Quad -- Matthias
void Interaction_Forces_M(bool psingle, TpKernel tkernel, bool floating, bool usedem, bool lamsps
	, TpDeltaSph tdelta, TpCellMode cellmode
	, float viscob, float viscof, unsigned bsbound, unsigned bsfluid
	, unsigned np, unsigned npb, unsigned npbok, tuint3 ncells
	, const int2 *begincell, tuint3 cellmin, const unsigned *dcell
	, const double2 *posxy, const double *posz, const float4 *pospress
	, const float4 *velrhop, const typecode *code, const unsigned *idp
	, const float *pore, const float *mass, const tsymatrix3f *tau
	, tsymatrix3f *taudot, tsymatrix3f *straindot, tsymatrix3f *spin	
	, float *viscdt, float* ar, float3 *ace, float *delta
	, TpShifting tshifting, float3 *shiftpos, float *shiftdetect
	, bool simulate2d, StKerInfo *kerinfo, JBlockSizeAuto *bsauto)
{
	if (tkernel == KERNEL_Wendland) {
		const TpKernel tker = KERNEL_Wendland;
		if (psingle) {
			const bool psingle = true;
			if (!floating)   Interaction_Forces_t1<psingle, tker, FTMODE_None>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
			else if (!usedem)Interaction_Forces_t1<psingle, tker, FTMODE_Sph>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
			else            Interaction_Forces_t1<psingle, tker, FTMODE_Dem>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
		}
		else {
			const bool psingle = false;
			if (!floating)   Interaction_Forces_t1<psingle, tker, FTMODE_None>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
			else if (!usedem)Interaction_Forces_t1<psingle, tker, FTMODE_Sph>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
			else            Interaction_Forces_t1<psingle, tker, FTMODE_Dem>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
		}
	}
	else if (tkernel == KERNEL_Gaussian) {
		const TpKernel tker = KERNEL_Gaussian;
		if (psingle) {
			const bool psingle = true;
			if (!floating)   Interaction_Forces_t1<psingle, tker, FTMODE_None>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
			else if (!usedem)Interaction_Forces_t1<psingle, tker, FTMODE_Sph>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
			else            Interaction_Forces_t1<psingle, tker, FTMODE_Dem>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
		}
		else {
			const bool psingle = false;
			if (!floating)   Interaction_Forces_t1<psingle, tker, FTMODE_None>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
			else if (!usedem)Interaction_Forces_t1<psingle, tker, FTMODE_Sph>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
			else            Interaction_Forces_t1<psingle, tker, FTMODE_Dem>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
		}
	}
	else if (tkernel == KERNEL_Cubic) {
		const TpKernel tker = KERNEL_Cubic;
		if (psingle) {
			const bool psingle = true;
			if (!floating)   Interaction_Forces_t1<psingle, tker, FTMODE_None>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
			else if (!usedem)Interaction_Forces_t1<psingle, tker, FTMODE_Sph>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
			else            Interaction_Forces_t1<psingle, tker, FTMODE_Dem>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
		}
		else {
			const bool psingle = false;
			if (!floating)   Interaction_Forces_t1<psingle, tker, FTMODE_None>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
			else if (!usedem)Interaction_Forces_t1<psingle, tker, FTMODE_Sph>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
			else            Interaction_Forces_t1<psingle, tker, FTMODE_Dem>(lamsps, tdelta, cellmode, viscob, viscof, bsbound, bsfluid, np, npb, npbok, ncells, begincell, cellmin, dcell, posxy, posz, pospress, velrhop, code, idp, pore, mass, tau, taudot, straindot, spin, viscdt, ar, ace, delta, tshifting, shiftpos, shiftdetect, simulate2d, kerinfo, bsauto);
		}
	}
}


//##############################################################################
//# Kernels for DEM interaction.
//# Kernels para interaccion DEM.
//##############################################################################
//------------------------------------------------------------------------------
/// DEM interaction of a particle with a set of particles. (Float-Float/Bound)
/// Realiza la interaccion DEM de una particula con un conjunto de ellas. (Float-Float/Bound)
//------------------------------------------------------------------------------
template<bool psingle> __device__ void KerInteractionForcesDemBox 
  (bool boundp2,const unsigned &pini,const unsigned &pfin
  ,const float4 *demdata,float dtforce
  ,const double2 *posxy,const double *posz,const float4 *pospress,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,double3 posdp1,float3 posp1,float3 velp1,typecode tavp1,float masstotp1,float taup1,float kfricp1,float restitup1
  ,float3 &acep1,float &demdtp1)
{
  for(int p2=pini;p2<pfin;p2++){
    const typecode codep2=code[p2];
    if(CODE_IsNotFluid(codep2) && tavp1!=CODE_GetTypeAndValue(codep2)){
      float drx,dry,drz;
      KerGetParticlesDr<psingle> (p2,posxy,posz,pospress,posdp1,posp1,drx,dry,drz);
      const float rr2=drx*drx+dry*dry+drz*drz;
      const float rad=sqrt(rr2);

      //-Computes maximum value of demdt.
      float4 demdatap2=demdata[CODE_GetTypeAndValue(codep2)];
      const float nu_mass=(boundp2? masstotp1/2: masstotp1*demdatap2.x/(masstotp1+demdatap2.x)); //-With boundary takes the actual mass of floating 1. | Con boundary toma la propia masa del floating 1.
      const float kn=4/(3*(taup1+demdatap2.y))*sqrt(CTE.dp/4); //-Generalized rigidity - Lemieux 2008.
      const float dvx=velp1.x-velrhop[p2].x, dvy=velp1.y-velrhop[p2].y, dvz=velp1.z-velrhop[p2].z; //vji
      const float nx=drx/rad, ny=dry/rad, nz=drz/rad; //-normal_ji             
      const float vn=dvx*nx+dvy*ny+dvz*nz; //-vji.nji    
      const float demvisc=0.2f/(3.21f*(pow(nu_mass/kn,0.4f)*pow(fabs(vn),-0.2f))/40.f);
      if(demdtp1<demvisc)demdtp1=demvisc;

      const float over_lap=1.0f*CTE.dp-rad; //-(ri+rj)-|dij|
      if(over_lap>0.0f){ //-Contact.
        //-Normal.
        const float eij=(restitup1+demdatap2.w)/2;
        const float gn=-(2.0f*log(eij)*sqrt(nu_mass*kn))/(sqrt(float(PI)+log(eij)*log(eij))); //-Generalized damping - Cummins 2010.
        //const float gn=0.08f*sqrt(nu_mass*sqrt(CTE.dp/2)/((taup1+demdatap2.y)/2)); //-generalized damping - Lemieux 2008.
        float rep=kn*pow(over_lap,1.5f);
        float fn=rep-gn*pow(over_lap,0.25f)*vn;                   
        acep1.x+=(fn*nx); acep1.y+=(fn*ny); acep1.z+=(fn*nz); //-Force is applied in the normal between the particles.
        //-Tangencial.
        float dvxt=dvx-vn*nx, dvyt=dvy-vn*ny, dvzt=dvz-vn*nz; //Vji_t
        float vt=sqrt(dvxt*dvxt + dvyt*dvyt + dvzt*dvzt);
        float tx=(vt!=0? dvxt/vt: 0), ty=(vt!=0? dvyt/vt: 0), tz=(vt!=0? dvzt/vt: 0); //-Tang vel unit vector.
        float ft_elast=2*(kn*dtforce-gn)*vt/7; //-Elastic frictional string -->  ft_elast=2*(kn*fdispl-gn*vt)/7; fdispl=dtforce*vt;
        const float kfric_ij=(kfricp1+demdatap2.z)/2;
        float ft=kfric_ij*fn*tanh(8*vt);  //-Coulomb.
        ft=(ft<ft_elast? ft: ft_elast);   //-Not above yield criteria, visco-elastic model.
        acep1.x+=(ft*tx); acep1.y+=(ft*ty); acep1.z+=(ft*tz);
      }
    }
  }
}

//------------------------------------------------------------------------------
/// Interaction between particles. Fluid/Float-Fluid/Float or Fluid/Float-Bound.
/// Includes artificial/laminar viscosity and normal/DEM floating bodies.
///
/// Realiza interaccion entre particulas. Fluid/Float-Fluid/Float or Fluid/Float-Bound
/// Incluye visco artificial/laminar y floatings normales/dem.
//------------------------------------------------------------------------------
template<bool psingle> __global__ void KerInteractionForcesDem
  (unsigned nfloat,int hdiv,int4 nc,unsigned cellfluid
  ,const int2 *begincell,int3 cellzero,const unsigned *dcell
  ,const unsigned *ftridp,const float4 *demdata,float dtforce
  ,const double2 *posxy,const double *posz,const float4 *pospress,const float4 *velrhop,const typecode *code,const unsigned *idp
  ,float *viscdt,float3 *ace)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<nfloat){
    const unsigned p1=ftridp[p]; //-Number of particle.
    if(p1!=UINT_MAX){
      float demdtp1=0;
      float3 acep1=make_float3(0,0,0);

      //-Obtains basic data of particle p1.
      double3 posdp1;
      float3 posp1,velp1;
      KerGetParticleData<psingle>(p1,posxy,posz,pospress,velrhop,velp1,posdp1,posp1);
      const typecode tavp1=CODE_GetTypeAndValue(code[p1]);
      float4 rdata=demdata[tavp1];
      const float masstotp1=rdata.x;
      const float taup1=rdata.y;
      const float kfricp1=rdata.z;
      const float restitup1=rdata.w;

      //-Obtains interaction limits.
      int cxini,cxfin,yini,yfin,zini,zfin;
      KerGetInteractionCells(dcell[p1],hdiv,nc,cellzero,cxini,cxfin,yini,yfin,zini,zfin);

      //-Interaction with boundaries.
      for(int z=zini;z<zfin;z++){
        int zmod=(nc.w)*z;
        for(int y=yini;y<yfin;y++){
          int ymod=zmod+nc.x*y;
          unsigned pini,pfin=0;
          for(int x=cxini;x<cxfin;x++){
            int2 cbeg=begincell[x+ymod];
            if(cbeg.y){
              if(!pfin)pini=cbeg.x;
              pfin=cbeg.y;
            }
          }
          if(pfin)KerInteractionForcesDemBox<psingle> (true ,pini,pfin,demdata,dtforce,posxy,posz,pospress,velrhop,code,idp,posdp1,posp1,velp1,tavp1,masstotp1,taup1,kfricp1,restitup1,acep1,demdtp1);
        }
      }

      //-Interaction with fluids.
      for(int z=zini;z<zfin;z++){
        int zmod=(nc.w)*z+cellfluid; //-The sum showing where fluid cells start. | Le suma donde empiezan las celdas de fluido.
        for(int y=yini;y<yfin;y++){
          int ymod=zmod+nc.x*y;
          unsigned pini,pfin=0;
          for(int x=cxini;x<cxfin;x++){
            int2 cbeg=begincell[x+ymod];
            if(cbeg.y){
              if(!pfin)pini=cbeg.x;
              pfin=cbeg.y;
            }
          }
          if(pfin)KerInteractionForcesDemBox<psingle> (false,pini,pfin,demdata,dtforce,posxy,posz,pospress,velrhop,code,idp,posdp1,posp1,velp1,tavp1,masstotp1,taup1,kfricp1,restitup1,acep1,demdtp1);
        }
      }
      //-Stores results.
      if(acep1.x || acep1.y || acep1.z || demdtp1){
        float3 r=ace[p1]; r.x+=acep1.x; r.y+=acep1.y; r.z+=acep1.z; ace[p1]=r;
        if(viscdt[p1]<demdtp1)viscdt[p1]=demdtp1;
      }
    }
  }
}

//==============================================================================
/// Collects kernel information.
//==============================================================================
template<bool psingle> void Interaction_ForcesDemT_KerInfo(StKerInfo *kerinfo)
{
#if CUDART_VERSION >= 6050
  {
    typedef void (*fun_ptr)(unsigned,int,int4,unsigned,const int2*,int3,const unsigned*,const unsigned*,const float4*,float,const double2*,const double*,const float4*,const float4*,const typecode*,const unsigned*,float*,float3*);
    fun_ptr ptr=&KerInteractionForcesDem<psingle>;
    int qblocksize=0,mingridsize=0;
    hipOccupancyMaxPotentialBlockSize(&mingridsize,&qblocksize,(void*)ptr,0,0);
    struct hipFuncAttributes attr;
    hipFuncGetAttributes(&attr,reinterpret_cast<const void*>((void*))ptr);
    kerinfo->forcesdem_bs=qblocksize;
    kerinfo->forcesdem_rg=attr.numRegs;
    kerinfo->forcesdem_bsmax=attr.maxThreadsPerBlock;
    //printf(">> KerInteractionForcesDem  blocksize:%u (%u)\n",qblocksize,0);
  }
  CheckErrorCuda("Error collecting kernel information.");
#endif
}

//==============================================================================
/// Interaction for the force computation.
/// Interaccion para el calculo de fuerzas.
//==============================================================================
template<bool psingle> void Interaction_ForcesDemT
  (TpCellMode cellmode,unsigned bsize
  ,unsigned nfloat,tuint3 ncells,const int2 *begincell,tuint3 cellmin,const unsigned *dcell
  ,const unsigned *ftridp,const float4 *demdata,float dtforce
  ,const double2 *posxy,const double *posz,const float4 *pospress,const float4 *velrhop
  ,const typecode *code,const unsigned *idp,float *viscdt,float3 *ace,StKerInfo *kerinfo)
{
  //-Collects kernel information.
  if(kerinfo)Interaction_ForcesDemT_KerInfo<psingle>(kerinfo);
  else{
    const int hdiv=(cellmode==CELLMODE_H? 2: 1);
    const int4 nc=make_int4(int(ncells.x),int(ncells.y),int(ncells.z),int(ncells.x*ncells.y));
    const unsigned cellfluid=nc.w*nc.z+1;
    const int3 cellzero=make_int3(cellmin.x,cellmin.y,cellmin.z);
    //-Interaction Fluid-Fluid & Fluid-Bound.
    if(nfloat){
      dim3 sgrid=GetGridSize(nfloat,bsize);
      KerInteractionForcesDem<psingle> <<<sgrid,bsize>>> (nfloat,hdiv,nc,cellfluid,begincell,cellzero,dcell,ftridp,demdata,dtforce,posxy,posz,pospress,velrhop,code,idp,viscdt,ace);
    }
  }
}
//==============================================================================
void Interaction_ForcesDem(bool psingle,TpCellMode cellmode,unsigned bsize
  ,unsigned nfloat,tuint3 ncells,const int2 *begincell,tuint3 cellmin,const unsigned *dcell
  ,const unsigned *ftridp,const float4 *demdata,float dtforce
  ,const double2 *posxy,const double *posz,const float4 *pospress,const float4 *velrhop
  ,const typecode *code,const unsigned *idp,float *viscdt,float3 *ace,StKerInfo *kerinfo)
{
  if(psingle)Interaction_ForcesDemT<true>  (cellmode,bsize,nfloat,ncells,begincell,cellmin,dcell,ftridp,demdata,dtforce,posxy,posz,pospress,velrhop,code,idp,viscdt,ace,kerinfo);
  else       Interaction_ForcesDemT<false> (cellmode,bsize,nfloat,ncells,begincell,cellmin,dcell,ftridp,demdata,dtforce,posxy,posz,pospress,velrhop,code,idp,viscdt,ace,kerinfo);
}


//##############################################################################
//# Kernels for Laminar+SPS.
//##############################################################################
//------------------------------------------------------------------------------
/// Computes sub-particle stress tensor (Tau) for SPS turbulence model.
//------------------------------------------------------------------------------
__global__ void KerComputeSpsTau(unsigned n,unsigned pini,float smag,float blin
  ,const float4 *velrhop,const float2 *gradvelff,float2 *tauff)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; 
  if(p<n){
    const unsigned p1=p+pini;
    float2 rr=gradvelff[p1*3];   const float grad_xx=rr.x,grad_xy=rr.y;
           rr=gradvelff[p1*3+1]; const float grad_xz=rr.x,grad_yy=rr.y;
           rr=gradvelff[p1*3+2]; const float grad_yz=rr.x,grad_zz=rr.y;
    const float pow1=grad_xx*grad_xx + grad_yy*grad_yy + grad_zz*grad_zz;
    const float prr= grad_xy*grad_xy + grad_xz*grad_xz + grad_yz*grad_yz + pow1+pow1;
    const float visc_sps=smag*sqrt(prr);
    const float div_u=grad_xx+grad_yy+grad_zz;
    const float sps_k=(2.0f/3.0f)*visc_sps*div_u;
    const float sps_blin=blin*prr;
    const float sumsps=-(sps_k+sps_blin);
    const float twovisc_sps=(visc_sps+visc_sps);
    float one_rho2=1.0f/velrhop[p1].w;
    //-Computes new values of tau[].
    const float tau_xx=one_rho2*(twovisc_sps*grad_xx +sumsps);
    const float tau_xy=one_rho2*(visc_sps   *grad_xy);
    tauff[p1*3]=make_float2(tau_xx,tau_xy);
    const float tau_xz=one_rho2*(visc_sps   *grad_xz);
    const float tau_yy=one_rho2*(twovisc_sps*grad_yy +sumsps);
    tauff[p1*3+1]=make_float2(tau_xz,tau_yy);
    const float tau_yz=one_rho2*(visc_sps   *grad_yz);
    const float tau_zz=one_rho2*(twovisc_sps*grad_zz +sumsps);
    tauff[p1*3+2]=make_float2(tau_yz,tau_zz);
  }
}

//==============================================================================
/// Computes sub-particle stress tensor (Tau) for SPS turbulence model.
//==============================================================================
void ComputeSpsTau(unsigned np,unsigned npb,float smag,float blin
  ,const float4 *velrhop,const tsymatrix3f *gradvelg,tsymatrix3f *tau)
{
  const unsigned npf=np-npb;
  if(npf){
    dim3 sgridf=GetGridSize(npf,SPHBSIZE);
    KerComputeSpsTau <<<sgridf,SPHBSIZE>>> (npf,npb,smag,blin,velrhop,(const float2*)gradvelg,(float2*)tau);
  }
}


//##############################################################################
//# Kernels for Delta-SPH.
//# Kernels para Delta-SPH.
//##############################################################################
//------------------------------------------------------------------------------
/// Adds value of delta[] to ar[] provided it is not FLT_MAX.
/// Añade valor de delta[] a ar[] siempre que no sea FLT_MAX.
//------------------------------------------------------------------------------
__global__ void KerAddDelta(unsigned n,const float *delta,float *ar)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    float rdelta=delta[p];
    if(rdelta!=FLT_MAX)ar[p]+=rdelta;
  }
}

//==============================================================================
/// Adds value of delta[] to ar[] provided it is not FLT_MAX.
/// Añade valor de delta[] a ar[] siempre que no sea FLT_MAX.
//==============================================================================
void AddDelta(unsigned n,const float *delta,float *ar){
  if(n){
    dim3 sgrid=GetGridSize(n,SPHBSIZE);
    KerAddDelta <<<sgrid,SPHBSIZE>>> (n,delta,ar);
  }
}


//##############################################################################
//# Kernels for Shifting.
//##############################################################################
//------------------------------------------------------------------------------
/// Computes final shifting for the particle position.
/// Calcula Shifting final para posicion de particulas.
//------------------------------------------------------------------------------
__global__ void KerRunShifting(unsigned n,unsigned pini,double dt
  ,float shiftcoef,float shifttfs,double coeftfs
  ,const float4 *velrhop,const float *shiftdetect,float3 *shiftpos)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p1=p+pini;
    const float4 rvel=velrhop[p1];
    const double vx=double(rvel.x);
    const double vy=double(rvel.y);
    const double vz=double(rvel.z);
    double umagn=double(shiftcoef)*double(CTE.h)*sqrt(vx*vx+vy*vy+vz*vz)*dt;
    if(shiftdetect){
      const float rdetect=shiftdetect[p1];
      if(rdetect<shifttfs)umagn=0;
      else umagn*=(double(rdetect)-shifttfs)/coeftfs;
    }
    float3 rshiftpos=shiftpos[p1];
    if(rshiftpos.x==FLT_MAX)umagn=0; //-Cancels shifting close to the boundaries. | Anula shifting por proximidad del contorno. 
    const float maxdist=0.1f*CTE.dp; //-Max shifting distance permitted (recommended).
    const float shiftdistx=float(double(rshiftpos.x)*umagn);
    const float shiftdisty=float(double(rshiftpos.y)*umagn);
    const float shiftdistz=float(double(rshiftpos.z)*umagn);
    rshiftpos.x=(shiftdistx<maxdist? shiftdistx: maxdist);
    rshiftpos.y=(shiftdisty<maxdist? shiftdisty: maxdist);
    rshiftpos.z=(shiftdistx<maxdist? shiftdistz: maxdist);
    shiftpos[p1]=rshiftpos;
  }
}

//==============================================================================
/// Computes final shifting for the particle position.
/// Calcula Shifting final para posicion de particulas.
//==============================================================================
void RunShifting(unsigned np,unsigned npb,double dt
  ,double shiftcoef,float shifttfs,double coeftfs
  ,const float4 *velrhop,const float *shiftdetect,float3 *shiftpos)
{
  const unsigned n=np-npb;
  if(n){
    dim3 sgrid=GetGridSize(n,SPHBSIZE);
    KerRunShifting <<<sgrid,SPHBSIZE>>> (n,npb,dt,shiftcoef,shifttfs,coeftfs,velrhop,shiftdetect,shiftpos);
  }
}


//##############################################################################
//# Kernels for ComputeStep (vel & rhop).
//# Kernels para ComputeStep (vel & rhop).
//##############################################################################
//------------------------------------------------------------------------------
/// Computes new values for Pos, Check, Vel and Ros (using Verlet).
/// The value of Vel always set to be reset.
///
/// Calcula nuevos valores de  Pos, Check, Vel y Rhop (usando Verlet).
/// El valor de Vel para bound siempre se pone a cero.
//------------------------------------------------------------------------------
template<bool floating,bool shift> __global__ void KerComputeStepVerlet
  (unsigned n,unsigned npb,float rhopoutmin,float rhopoutmax
  ,const float4 *velrhop1,const float4 *velrhop2
  ,const float *ar,const float3 *ace,const float3 *shiftpos
  ,double dt,double dt205,double dt2
  ,double2 *movxy,double *movz,typecode *code,float4 *velrhopnew)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    if(p<npb){ //-Particles: Fixed & Moving.
      float rrhop=float(double(velrhop2[p].w)+dt2*ar[p]);
      rrhop=(rrhop<CTE.rhopzero? CTE.rhopzero: rrhop); //-To prevent absorption of fluid particles by boundaries. | Evita q las boundary absorvan a las fluidas.
      velrhopnew[p]=make_float4(0,0,0,rrhop);
    }
    else{ //-Particles: Floating & Fluid.
      //-Updates density.
      float4 rvelrhop2=velrhop2[p];
      rvelrhop2.w=float(double(rvelrhop2.w)+dt2*ar[p]);
      float4 rvel1=velrhop1[p];
      if(!floating || CODE_IsFluid(code[p])){ //-Particles: Fluid.
        //-Checks rhop limits.
        if(rvelrhop2.w<rhopoutmin||rvelrhop2.w>rhopoutmax){ //-Only brands as excluded normal particles (not periodic). | Solo marca como excluidas las normales (no periodicas).
          const typecode rcode=code[p];
          if(CODE_IsNormal(rcode))code[p]=CODE_SetOutRhop(rcode);
        }
        //-Computes and stores position displacement.
        const float3 race=ace[p];
        double dx=double(rvel1.x)*dt + double(race.x)*dt205;
        double dy=double(rvel1.y)*dt + double(race.y)*dt205;
        double dz=double(rvel1.z)*dt + double(race.z)*dt205;
        if(shift){
          const float3 rshiftpos=shiftpos[p];
          dx+=double(rshiftpos.x);
          dy+=double(rshiftpos.y);
          dz+=double(rshiftpos.z);
        }
        movxy[p]=make_double2(dx,dy);
        movz[p]=dz;
        //-Updates velocity.
        rvelrhop2.x=float(double(rvelrhop2.x)+double(race.x)*dt2);
        rvelrhop2.y=float(double(rvelrhop2.y)+double(race.y)*dt2);
        rvelrhop2.z=float(double(rvelrhop2.z)+double(race.z)*dt2);
        velrhopnew[p]=rvelrhop2;
      }
      else{ //-Particles: Floating.
        rvel1.w=(rvelrhop2.w<CTE.rhopzero? CTE.rhopzero: rvelrhop2.w); //-To prevent absorption of fluid particles by boundaries. | Evita q las floating absorvan a las fluidas.
        velrhopnew[p]=rvel1;
      }
    }
  }
}

	//==============================================================================
	/// Updates particles according to forces and dt using Verlet. 
	/// Actualizacion de particulas segun fuerzas y dt usando Verlet.
	//==============================================================================
	void ComputeStepVerlet(bool floating,bool shift,unsigned np,unsigned npb
	  ,const float4 *velrhop1,const float4 *velrhop2
	  ,const float *ar,const float3 *ace,const float3 *shiftpos
	  ,double dt,double dt2,float rhopoutmin,float rhopoutmax
	  ,typecode *code,double2 *movxy,double *movz,float4 *velrhopnew)
	{
	  double dt205=(0.5*dt*dt);
	  if(np){
		dim3 sgrid=GetGridSize(np,SPHBSIZE);
		if(shift){    const bool shift=true;
		  if(floating)KerComputeStepVerlet<true,shift>  <<<sgrid,SPHBSIZE>>> (np,npb,rhopoutmin,rhopoutmax,velrhop1,velrhop2,ar,ace,shiftpos,dt,dt205,dt2,movxy,movz,code,velrhopnew);
		  else        KerComputeStepVerlet<false,shift> <<<sgrid,SPHBSIZE>>> (np,npb,rhopoutmin,rhopoutmax,velrhop1,velrhop2,ar,ace,shiftpos,dt,dt205,dt2,movxy,movz,code,velrhopnew);
		}else{        const bool shift=false;
		  if(floating)KerComputeStepVerlet<true,shift>  <<<sgrid,SPHBSIZE>>> (np,npb,rhopoutmin,rhopoutmax,velrhop1,velrhop2,ar,ace,shiftpos,dt,dt205,dt2,movxy,movz,code,velrhopnew);
		  else        KerComputeStepVerlet<false,shift> <<<sgrid,SPHBSIZE>>> (np,npb,rhopoutmin,rhopoutmax,velrhop1,velrhop2,ar,ace,shiftpos,dt,dt205,dt2,movxy,movz,code,velrhopnew);
		}
	  }
	}

//------------------------------------------------------------------------------
/// Computes new values for Pos, Check, Vel and Ros (used with Symplectic-Predictor).
/// Calcula los nuevos valores de Pos, Vel y Rhop (usando para Symplectic-Predictor).
//------------------------------------------------------------------------------
template<bool floating,bool shift> __global__ void KerComputeStepSymplecticPre
  (unsigned n,unsigned npb
  ,const float4 *velrhoppre,const float *ar,const float3 *ace,const float3 *shiftpos
  ,double dtm,float rhopoutmin,float rhopoutmax
  ,typecode *code,double2 *movxy,double *movz,float4 *velrhop)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    if(p<npb){ //-Particles: Fixed & Moving.
      float4 rvelrhop=velrhoppre[p];
      rvelrhop.w=float(double(rvelrhop.w)+dtm*ar[p]);
      rvelrhop.w=(rvelrhop.w<CTE.rhopzero? CTE.rhopzero: rvelrhop.w); //-To prevent absorption of fluid particles by boundaries. | Evita que las boundary absorvan a las fluidas.
      velrhop[p]=rvelrhop;
    }
    else{ //-Particles: Floating & Fluid.
      //-Updates density.
      float4 rvelrhop=velrhoppre[p];
      rvelrhop.w=float(double(rvelrhop.w)+dtm*ar[p]);
      if(!floating || CODE_IsFluid(code[p])){ //-Particles: Fluid.
        //-Checks rhop limits.
        if(rvelrhop.w<rhopoutmin||rvelrhop.w>rhopoutmax){//-Only brands as excluded normal particles (not periodic). | Solo marca como excluidas las normales (no periodicas).
          const typecode rcode=code[p];
          if(CODE_IsNormal(rcode))code[p]=CODE_SetOutRhop(rcode);
        }
        //-Computes and stores position displacement.
        double dx=double(rvelrhop.x)*dtm;
        double dy=double(rvelrhop.y)*dtm;
        double dz=double(rvelrhop.z)*dtm;
        if(shift){
          const float3 rshiftpos=shiftpos[p];
          dx+=double(rshiftpos.x);
          dy+=double(rshiftpos.y);
          dz+=double(rshiftpos.z);
        }
        movxy[p]=make_double2(dx,dy);
        movz[p]=dz;
        //-Updates velocity.
        const float3 race=ace[p];
        rvelrhop.x=float(double(rvelrhop.x)+double(race.x)*dtm);
        rvelrhop.y=float(double(rvelrhop.y)+double(race.y)*dtm);
        rvelrhop.z=float(double(rvelrhop.z)+double(race.z)*dtm);
      }
      else{ //-Particles: Floating.
        rvelrhop.w=(rvelrhop.w<CTE.rhopzero? CTE.rhopzero: rvelrhop.w); //-To prevent absorption of fluid particles by boundaries. | Evita q las floating absorvan a las fluidas.
      }
      //-Stores new velocity and density.
      velrhop[p]=rvelrhop;
    }
  }
}

//==============================================================================
/// Updates particles using Symplectic-Predictor.
/// Actualizacion de particulas usando Symplectic-Predictor.
//==============================================================================   
void ComputeStepSymplecticPre(bool floating,bool shift,unsigned np,unsigned npb
  ,const float4 *velrhoppre,const float *ar,const float3 *ace,const float3 *shiftpos
  ,double dtm,float rhopoutmin,float rhopoutmax
  ,typecode *code,double2 *movxy,double *movz,float4 *velrhop)
{
  if(np){
    dim3 sgrid=GetGridSize(np,SPHBSIZE);
    if(shift){    const bool shift=false; //-We strongly recommend running the shifting correction only for the corrector. If you want to re-enable shifting in the predictor, change the value here to "true".
      if(floating)KerComputeStepSymplecticPre<true ,shift> <<<sgrid,SPHBSIZE>>> (np,npb,velrhoppre,ar,ace,shiftpos,dtm,rhopoutmin,rhopoutmax,code,movxy,movz,velrhop);
      else        KerComputeStepSymplecticPre<false,shift> <<<sgrid,SPHBSIZE>>> (np,npb,velrhoppre,ar,ace,shiftpos,dtm,rhopoutmin,rhopoutmax,code,movxy,movz,velrhop);
    }else{        const bool shift=false;
      if(floating)KerComputeStepSymplecticPre<true ,shift> <<<sgrid,SPHBSIZE>>> (np,npb,velrhoppre,ar,ace,shiftpos,dtm,rhopoutmin,rhopoutmax,code,movxy,movz,velrhop);
      else        KerComputeStepSymplecticPre<false,shift> <<<sgrid,SPHBSIZE>>> (np,npb,velrhoppre,ar,ace,shiftpos,dtm,rhopoutmin,rhopoutmax,code,movxy,movz,velrhop);
    }
  }
}

//------------------------------------------------------------------------------
/// Computes new values for Pos, Check, Vel and Ros (using Verlet).
/// The value of Vel always set to be reset.
///
/// Calcula los nuevos valores de Pos, Vel y Rhop (usandopara Symplectic-Corrector).
/// Pone vel de contorno a cero.
//------------------------------------------------------------------------------
template<bool floating,bool shift> __global__ void KerComputeStepSymplecticCor
  (unsigned n,unsigned npb
  ,const float4 *velrhoppre,const float *ar,const float3 *ace,const float3 *shiftpos
  ,double dtm,double dt,float rhopoutmin,float rhopoutmax
  ,typecode *code,double2 *movxy,double *movz,float4 *velrhop)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    if(p<npb){ //-Particles: Fixed & Moving.
      double epsilon_rdot=(-double(ar[p])/double(velrhop[p].w))*dt;
      float rrhop=float(double(velrhoppre[p].w) * (2.-epsilon_rdot)/(2.+epsilon_rdot));
      rrhop=(rrhop<CTE.rhopzero? CTE.rhopzero: rrhop); //-To prevent absorption of fluid particles by boundaries. | Evita q las boundary absorvan a las fluidas.
      velrhop[p]=make_float4(0,0,0,rrhop);
    }
    else{ //-Particles: Floating & Fluid.
      //-Updates density.
      double epsilon_rdot=(-double(ar[p])/double(velrhop[p].w))*dt;
      float4 rvelrhop=velrhoppre[p];
      rvelrhop.w=float(double(rvelrhop.w) * (2.-epsilon_rdot)/(2.+epsilon_rdot));
      if(!floating || CODE_IsFluid(code[p])){//-Particles: Fluid.
        float4 rvelp=rvelrhop;
        //-Updates velocity.
        float3 race=ace[p];
        rvelrhop.x=float(double(rvelrhop.x)+double(race.x)*dt);
        rvelrhop.y=float(double(rvelrhop.y)+double(race.y)*dt);
        rvelrhop.z=float(double(rvelrhop.z)+double(race.z)*dt);
        //-Checks rhop limits.
        if(rvelrhop.w<rhopoutmin||rvelrhop.w>rhopoutmax){//-Only brands as excluded normal particles (not periodic). | Solo marca como excluidas las normales (no periodicas).
          const typecode rcode=code[p];
          if(CODE_IsNormal(rcode))code[p]=CODE_SetOutRhop(rcode);
        }
        //-Computes and stores position displacement.
        double dx=(double(rvelp.x)+double(rvelrhop.x))*dtm;
        double dy=(double(rvelp.y)+double(rvelrhop.y))*dtm;
        double dz=(double(rvelp.z)+double(rvelrhop.z))*dtm;
        if(shift){
          const float3 rshiftpos=shiftpos[p];
          dx+=double(rshiftpos.x);
          dy+=double(rshiftpos.y);
          dz+=double(rshiftpos.z);
        }
        movxy[p]=make_double2(dx,dy);
        movz[p]=dz;
      }
      else{ //-Particles: Floating.
        rvelrhop.w=(rvelrhop.w<CTE.rhopzero? CTE.rhopzero: rvelrhop.w); //-To prevent absorption of fluid particles by boundaries. | Evita q las floating absorvan a las fluidas.
      }
      //-Stores new velocity and density.
      velrhop[p]=rvelrhop;
    }
  }
}

//==============================================================================
/// Updates particles using Symplectic-Corrector.
/// Actualizacion de particulas usando Symplectic-Corrector.
//==============================================================================   
void ComputeStepSymplecticCor(bool floating,bool shift,unsigned np,unsigned npb
  ,const float4 *velrhoppre,const float *ar,const float3 *ace,const float3 *shiftpos
  ,double dtm,double dt,float rhopoutmin,float rhopoutmax
  ,typecode *code,double2 *movxy,double *movz,float4 *velrhop)
{
  if(np){
    dim3 sgrid=GetGridSize(np,SPHBSIZE);
    if(shift){    const bool shift=true;
      if(floating)KerComputeStepSymplecticCor<true,shift>  <<<sgrid,SPHBSIZE>>> (np,npb,velrhoppre,ar,ace,shiftpos,dtm,dt,rhopoutmin,rhopoutmax,code,movxy,movz,velrhop);
      else        KerComputeStepSymplecticCor<false,shift> <<<sgrid,SPHBSIZE>>> (np,npb,velrhoppre,ar,ace,shiftpos,dtm,dt,rhopoutmin,rhopoutmax,code,movxy,movz,velrhop);
    }else{        const bool shift=false;
      if(floating)KerComputeStepSymplecticCor<true,shift>  <<<sgrid,SPHBSIZE>>> (np,npb,velrhoppre,ar,ace,shiftpos,dtm,dt,rhopoutmin,rhopoutmax,code,movxy,movz,velrhop);
      else        KerComputeStepSymplecticCor<false,shift> <<<sgrid,SPHBSIZE>>> (np,npb,velrhoppre,ar,ace,shiftpos,dtm,dt,rhopoutmin,rhopoutmax,code,movxy,movz,velrhop);
    }
  }
}


//##############################################################################
//# Kernels para ComputeStep (position)
//# Kernels for ComputeStep (position)
//##############################################################################
//------------------------------------------------------------------------------
/// Updates pos, dcell and code from the indicated displacement.
/// The code may be CODE_OUTRHOP because in ComputeStepVerlet / Symplectic this is evaluated
/// and is executed before ComputeStepPos.
/// Checks limits depending on maprealposmin and maprealsize, this is valid 
/// for single-GPU because maprealpos and domrealpos are equal. For multi-gpu it is
/// important to mark particles that leave the domain without leaving the map.
///
/// Actualiza pos, dcell y code a partir del desplazamiento indicado.
/// Code puede ser CODE_OUTRHOP pq en ComputeStepVerlet/Symplectic se evalua esto 
/// y se ejecuta antes que ComputeStepPos.
/// Comprueba los limites en funcion de maprealposmin y maprealsize esto es valido
/// para single-gpu pq domrealpos y maprealpos son iguales. Para multi-gpu seria 
/// necesario marcar las particulas q salgan del dominio sin salir del mapa.
//------------------------------------------------------------------------------
template<bool periactive> __device__ void KerUpdatePos
  (double2 rxy,double rz,double movx,double movy,double movz
  ,bool outrhop,unsigned p,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  //-Checks validity of displacement. | Comprueba validez del desplazamiento.
  bool outmove=(fmaxf(fabsf(float(movx)),fmaxf(fabsf(float(movy)),fabsf(float(movz))))>CTE.movlimit);
  //-Applies diplacement.
  double3 rpos=make_double3(rxy.x,rxy.y,rz);
  rpos.x+=movx; rpos.y+=movy; rpos.z+=movz;
  //-Checks limits of real domain. | Comprueba limites del dominio reales.
  double dx=rpos.x-CTE.maprealposminx;
  double dy=rpos.y-CTE.maprealposminy;
  double dz=rpos.z-CTE.maprealposminz;
  bool out=(dx!=dx || dy!=dy || dz!=dz || dx<0 || dy<0 || dz<0 || dx>=CTE.maprealsizex || dy>=CTE.maprealsizey || dz>=CTE.maprealsizez);
  if(periactive && out){
    bool xperi=(CTE.periactive&1),yperi=(CTE.periactive&2),zperi=(CTE.periactive&4);
    if(xperi){
      if(dx<0)                { dx-=CTE.xperincx; dy-=CTE.xperincy; dz-=CTE.xperincz; }
      if(dx>=CTE.maprealsizex){ dx+=CTE.xperincx; dy+=CTE.xperincy; dz+=CTE.xperincz; }
    }
    if(yperi){
      if(dy<0)                { dx-=CTE.yperincx; dy-=CTE.yperincy; dz-=CTE.yperincz; }
      if(dy>=CTE.maprealsizey){ dx+=CTE.yperincx; dy+=CTE.yperincy; dz+=CTE.yperincz; }
    }
    if(zperi){
      if(dz<0)                { dx-=CTE.zperincx; dy-=CTE.zperincy; dz-=CTE.zperincz; }
      if(dz>=CTE.maprealsizez){ dx+=CTE.zperincx; dy+=CTE.zperincy; dz+=CTE.zperincz; }
    }
    bool outx=!xperi && (dx<0 || dx>=CTE.maprealsizex);
    bool outy=!yperi && (dy<0 || dy>=CTE.maprealsizey);
    bool outz=!zperi && (dz<0 || dz>=CTE.maprealsizez);
    out=(outx||outy||outz);
    rpos=make_double3(dx+CTE.maprealposminx,dy+CTE.maprealposminy,dz+CTE.maprealposminz);
  }
  //-Stores updated position.
  posxy[p]=make_double2(rpos.x,rpos.y);
  posz[p]=rpos.z;
  //-Stores cell and check. | Guarda celda y check.
  if(outrhop || outmove || out){//-Particle out. Only brands as excluded normal particles (not periodic). | Particle out. Solo las particulas normales (no periodicas) se pueden marcar como excluidas.
    typecode rcode=code[p];
    if(out)rcode=CODE_SetOutPos(rcode);
    else if(outrhop)rcode=CODE_SetOutRhop(rcode);
    else rcode=CODE_SetOutMove(rcode);
    code[p]=rcode;
    dcell[p]=0xFFFFFFFF;
  }
  else{//-Particle in.
    if(periactive){
      dx=rpos.x-CTE.domposminx;
      dy=rpos.y-CTE.domposminy;
      dz=rpos.z-CTE.domposminz;
    }
    unsigned cx=unsigned(dx/CTE.scell),cy=unsigned(dy/CTE.scell),cz=unsigned(dz/CTE.scell);
    dcell[p]=PC__Cell(CTE.cellcode,cx,cy,cz);
  }
}

//------------------------------------------------------------------------------
/// Returns the corrected position after applying periodic conditions.
/// Devuelve la posicion corregida tras aplicar condiciones periodicas.
//------------------------------------------------------------------------------
__device__ double3 KerUpdatePeriodicPos(double3 ps)
{
  double dx=ps.x-CTE.maprealposminx;
  double dy=ps.y-CTE.maprealposminy;
  double dz=ps.z-CTE.maprealposminz;
  const bool out=(dx!=dx || dy!=dy || dz!=dz || dx<0 || dy<0 || dz<0 || dx>=CTE.maprealsizex || dy>=CTE.maprealsizey || dz>=CTE.maprealsizez);
  //-Adjusts position according to periodic conditions and rechecks domain limits.
  //-Ajusta posicion segun condiciones periodicas y vuelve a comprobar los limites del dominio.
  if(out){
    bool xperi=(CTE.periactive&1),yperi=(CTE.periactive&2),zperi=(CTE.periactive&4);
    if(xperi){
      if(dx<0)                { dx-=CTE.xperincx; dy-=CTE.xperincy; dz-=CTE.xperincz; }
      if(dx>=CTE.maprealsizex){ dx+=CTE.xperincx; dy+=CTE.xperincy; dz+=CTE.xperincz; }
    }
    if(yperi){
      if(dy<0)                { dx-=CTE.yperincx; dy-=CTE.yperincy; dz-=CTE.yperincz; }
      if(dy>=CTE.maprealsizey){ dx+=CTE.yperincx; dy+=CTE.yperincy; dz+=CTE.yperincz; }
    }
    if(zperi){
      if(dz<0)                { dx-=CTE.zperincx; dy-=CTE.zperincy; dz-=CTE.zperincz; }
      if(dz>=CTE.maprealsizez){ dx+=CTE.zperincx; dy+=CTE.zperincy; dz+=CTE.zperincz; }
    }
    ps=make_double3(dx+CTE.maprealposminx,dy+CTE.maprealposminy,dz+CTE.maprealposminz);
  }
  return(ps);
}

//------------------------------------------------------------------------------
/// Updates particle position according to displacement.
/// Actualizacion de posicion de particulas segun desplazamiento.
//------------------------------------------------------------------------------
template<bool periactive,bool floating> __global__ void KerComputeStepPos(unsigned n,unsigned pini
  ,const double2 *movxy,const double *movz
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  unsigned pt=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(pt<n){
    unsigned p=pt+pini;
    const typecode rcode=code[p];
    const bool outrhop=CODE_IsOutRhop(rcode);
    const bool fluid=(!floating || CODE_IsFluid(rcode));
    const bool normal=(!periactive || outrhop || CODE_IsNormal(rcode));
    if(normal && fluid){ //-Does not apply to periodic or floating particles. | No se aplica a particulas periodicas o floating.
      const double2 rmovxy=movxy[p];
      KerUpdatePos<periactive>(posxy[p],posz[p],rmovxy.x,rmovxy.y,movz[p],outrhop,p,posxy,posz,dcell,code);
    }
    //-In case of floating maintains the original position.
    //-En caso de floating mantiene la posicion original.
  }
}

//==============================================================================
/// Updates particle position according to displacement.
/// Actualizacion de posicion de particulas segun desplazamiento.
//==============================================================================
void ComputeStepPos(byte periactive,bool floating,unsigned np,unsigned npb
  ,const double2 *movxy,const double *movz
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  const unsigned pini=npb;
  const unsigned npf=np-pini;
  if(npf){
    dim3 sgrid=GetGridSize(npf,SPHBSIZE);
    if(periactive){ const bool peri=true;
      if(floating)KerComputeStepPos<peri,true>  <<<sgrid,SPHBSIZE>>> (npf,pini,movxy,movz,posxy,posz,dcell,code);
      else        KerComputeStepPos<peri,false> <<<sgrid,SPHBSIZE>>> (npf,pini,movxy,movz,posxy,posz,dcell,code);
    }
    else{ const bool peri=false;
      if(floating)KerComputeStepPos<peri,true>  <<<sgrid,SPHBSIZE>>> (npf,pini,movxy,movz,posxy,posz,dcell,code);
      else        KerComputeStepPos<peri,false> <<<sgrid,SPHBSIZE>>> (npf,pini,movxy,movz,posxy,posz,dcell,code);
    }
  }
}

//------------------------------------------------------------------------------
/// Updates particle position according to displacement.
/// Actualizacion de posicion de particulas segun desplazamiento.
//------------------------------------------------------------------------------
template<bool periactive,bool floating> __global__ void KerComputeStepPos2(unsigned n,unsigned pini
  ,const double2 *posxypre,const double *poszpre,const double2 *movxy,const double *movz
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  unsigned pt=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(pt<n){
    unsigned p=pt+pini;
    const typecode rcode=code[p];
    const bool outrhop=CODE_IsOutRhop(rcode);
    const bool fluid=(!floating || CODE_IsFluid(rcode));
    const bool normal=(!periactive || outrhop || CODE_IsNormal(rcode));
    if(normal){//-Does not apply to periodic particles. | No se aplica a particulas periodicas
      if(fluid){//-Only applied for fluid displacement. | Solo se aplica desplazamiento al fluido.
        const double2 rmovxy=movxy[p];
        KerUpdatePos<periactive>(posxypre[p],poszpre[p],rmovxy.x,rmovxy.y,movz[p],outrhop,p,posxy,posz,dcell,code);
      }
      else{
        posxy[p]=posxypre[p];
        posz[p]=poszpre[p];
      }
    }
  }
}

//==============================================================================
/// Updates particle position according to displacement.
/// Actualizacion de posicion de particulas segun desplazamiento.
//==============================================================================
void ComputeStepPos2(byte periactive,bool floating,unsigned np,unsigned npb
  ,const double2 *posxypre,const double *poszpre,const double2 *movxy,const double *movz
  ,double2 *posxy,double *posz,unsigned *dcell,typecode *code)
{
  const unsigned pini=npb;
  const unsigned npf=np-pini;
  if(npf){
    dim3 sgrid=GetGridSize(npf,SPHBSIZE);
    if(periactive){ const bool peri=true;
      if(floating)KerComputeStepPos2<peri,true>  <<<sgrid,SPHBSIZE>>> (npf,pini,posxypre,poszpre,movxy,movz,posxy,posz,dcell,code);
      else        KerComputeStepPos2<peri,false> <<<sgrid,SPHBSIZE>>> (npf,pini,posxypre,poszpre,movxy,movz,posxy,posz,dcell,code);
    }
    else{ const bool peri=false;
      if(floating)KerComputeStepPos2<peri,true>  <<<sgrid,SPHBSIZE>>> (npf,pini,posxypre,poszpre,movxy,movz,posxy,posz,dcell,code);
      else        KerComputeStepPos2<peri,false> <<<sgrid,SPHBSIZE>>> (npf,pini,posxypre,poszpre,movxy,movz,posxy,posz,dcell,code);
    }
  }
}


//##############################################################################
//# Kernels for motion.
//# Kernels para Motion
//##############################################################################
//------------------------------------------------------------------------------
/// Computes for a range of particles, their position according to idp[].
/// Calcula para un rango de particulas calcula su posicion segun idp[].
//------------------------------------------------------------------------------
__global__ void KerCalcRidp(unsigned n,unsigned ini,unsigned idini,unsigned idfin,const typecode *code,const unsigned *idp,unsigned *ridp)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    p+=ini;
    unsigned id=idp[p];
    if(idini<=id && id<idfin){
      if(CODE_IsNormal(code[p]))ridp[id-idini]=p;
    }
  }
}
//------------------------------------------------------------------------------
__global__ void KerCalcRidp(unsigned n,unsigned ini,unsigned idini,unsigned idfin,const unsigned *idp,unsigned *ridp)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    p+=ini;
    const unsigned id=idp[p];
    if(idini<=id && id<idfin)ridp[id-idini]=p;
  }
}

//==============================================================================
/// Calculate particle position according to idp[]. When it does not find UINT_MAX.
/// When periactive is false it means there are no duplicate particles (periodic)
/// and all are CODE_NORMAL.
///
/// Calcula posicion de particulas segun idp[]. Cuando no la encuentra es UINT_MAX.
/// Cuando periactive es False sumpone que no hay particulas duplicadas (periodicas)
/// y todas son CODE_NORMAL.
//==============================================================================
void CalcRidp(bool periactive,unsigned np,unsigned pini,unsigned idini,unsigned idfin,const typecode *code,const unsigned *idp,unsigned *ridp){
  //-Assigns values UINT_MAX
  const unsigned nsel=idfin-idini;
  hipMemset(ridp,255,sizeof(unsigned)*nsel); 
  //-Computes position according to id. | Calcula posicion segun id.
  if(np){
    dim3 sgrid=GetGridSize(np,SPHBSIZE);
    if(periactive)KerCalcRidp <<<sgrid,SPHBSIZE>>> (np,pini,idini,idfin,code,idp,ridp);
    else          KerCalcRidp <<<sgrid,SPHBSIZE>>> (np,pini,idini,idfin,idp,ridp);
  }
}

//------------------------------------------------------------------------------
/// Applies a linear movement to a set of particles.
/// Aplica un movimiento lineal a un conjunto de particulas.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerMoveLinBound(unsigned n,unsigned ini,double3 mvpos,float3 mvvel
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    int pid=ridpmv[p+ini];
    if(pid>=0){
      //-Computes displacement and updates position.
      KerUpdatePos<periactive>(posxy[pid],posz[pid],mvpos.x,mvpos.y,mvpos.z,false,pid,posxy,posz,dcell,code);
      //-Computes velocity.
      velrhop[pid]=make_float4(mvvel.x,mvvel.y,mvvel.z,velrhop[pid].w);
    }
  }
}

//==============================================================================
/// Applies a linear movement to a set of particles.
/// Aplica un movimiento lineal a un conjunto de particulas.
//==============================================================================
void MoveLinBound(byte periactive,unsigned np,unsigned ini,tdouble3 mvpos,tfloat3 mvvel
  ,const unsigned *ridp,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  dim3 sgrid=GetGridSize(np,SPHBSIZE);
  if(periactive)KerMoveLinBound<true>  <<<sgrid,SPHBSIZE>>> (np,ini,Double3(mvpos),Float3(mvvel),ridp,posxy,posz,dcell,velrhop,code);
  else          KerMoveLinBound<false> <<<sgrid,SPHBSIZE>>> (np,ini,Double3(mvpos),Float3(mvvel),ridp,posxy,posz,dcell,velrhop,code);
}



//------------------------------------------------------------------------------
/// Applies a linear movement to a set of particles.
/// Aplica un movimiento matricial a un conjunto de particulas.
//------------------------------------------------------------------------------
template<bool periactive,bool simulate2d> __global__ void KerMoveMatBound(unsigned n,unsigned ini,tmatrix4d m,double dt
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    int pid=ridpmv[p+ini];
    if(pid>=0){
      double2 rxy=posxy[pid];
      double3 rpos=make_double3(rxy.x,rxy.y,posz[pid]);
      //-Computes new position.
      double3 rpos2;
      rpos2.x= rpos.x*m.a11 + rpos.y*m.a12 + rpos.z*m.a13 + m.a14;
      rpos2.y= rpos.x*m.a21 + rpos.y*m.a22 + rpos.z*m.a23 + m.a24;
      rpos2.z= rpos.x*m.a31 + rpos.y*m.a32 + rpos.z*m.a33 + m.a34;
      if(simulate2d)rpos2.y=rpos.y;
      //-Computes displacement and updates position.
      const double dx=rpos2.x-rpos.x;
      const double dy=rpos2.y-rpos.y;
      const double dz=rpos2.z-rpos.z;
      KerUpdatePos<periactive>(make_double2(rpos.x,rpos.y),rpos.z,dx,dy,dz,false,pid,posxy,posz,dcell,code);
      //-Computes velocity.
      velrhop[pid]=make_float4(float(dx/dt),float(dy/dt),float(dz/dt),velrhop[pid].w);
    }
  }
}

//==============================================================================
/// Applies a linear movement to a set of particles.
/// Aplica un movimiento matricial a un conjunto de particulas.
//==============================================================================
void MoveMatBound(byte periactive,bool simulate2d,unsigned np,unsigned ini,tmatrix4d m,double dt
  ,const unsigned *ridpmv,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  dim3 sgrid=GetGridSize(np,SPHBSIZE);
  if(periactive){ const bool peri=true;
    if(simulate2d)KerMoveMatBound<peri,true>  <<<sgrid,SPHBSIZE>>> (np,ini,m,dt,ridpmv,posxy,posz,dcell,velrhop,code);
    else          KerMoveMatBound<peri,false> <<<sgrid,SPHBSIZE>>> (np,ini,m,dt,ridpmv,posxy,posz,dcell,velrhop,code);
  }
  else{ const bool peri=false;
    if(simulate2d)KerMoveMatBound<peri,true>  <<<sgrid,SPHBSIZE>>> (np,ini,m,dt,ridpmv,posxy,posz,dcell,velrhop,code);
    else          KerMoveMatBound<peri,false> <<<sgrid,SPHBSIZE>>> (np,ini,m,dt,ridpmv,posxy,posz,dcell,velrhop,code);
  }
}

//##############################################################################
//# Kernels for Floating bodies.
//##############################################################################
//==============================================================================
/// Computes distance between floating and centre particles according to periodic conditions.
/// Calcula distancia entre pariculas floating y centro segun condiciones periodicas.
//==============================================================================
template<bool periactive> __device__ void KerFtPeriodicDist(double px,double py,double pz,double cenx,double ceny,double cenz,float radius,float &dx,float &dy,float &dz){
  if(periactive){
    double ddx=px-cenx;
    double ddy=py-ceny;
    double ddz=pz-cenz;
    const unsigned peri=CTE.periactive;
    if((peri&1) && fabs(ddx)>radius){
      if(ddx>0){ ddx+=CTE.xperincx; ddy+=CTE.xperincy; ddz+=CTE.xperincz; }
      else{      ddx-=CTE.xperincx; ddy-=CTE.xperincy; ddz-=CTE.xperincz; }
    }
    if((peri&2) && fabs(ddy)>radius){
      if(ddy>0){ ddx+=CTE.yperincx; ddy+=CTE.yperincy; ddz+=CTE.yperincz; }
      else{      ddx-=CTE.yperincx; ddy-=CTE.yperincy; ddz-=CTE.yperincz; }
    }
    if((peri&4) && fabs(ddz)>radius){
      if(ddz>0){ ddx+=CTE.zperincx; ddy+=CTE.zperincy; ddz+=CTE.zperincz; }
      else{      ddx-=CTE.zperincx; ddy-=CTE.zperincy; ddz-=CTE.zperincz; }
    }
    dx=float(ddx);
    dy=float(ddy);
    dz=float(ddz);
  }
  else{
    dx=float(px-cenx);
    dy=float(py-ceny);
    dz=float(pz-cenz);
  }
}

//------------------------------------------------------------------------------
/// Calculate summation: face, fomegaace in ftoforcessum[].
/// Calcula suma de face y fomegaace a partir de particulas floating en ftoforcessum[].
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerFtCalcForcesSum( //fdata={pini,np,radius,mass}
  float3 gravity,const float4 *ftodata,const double3 *ftocenter,const unsigned *ftridp
  ,const double2 *posxy,const double *posz,const float3 *ace
  ,float3 *ftoforcessum)
{
  extern __shared__ float rfacex[];
  float *rfacey=rfacex+blockDim.x;
  float *rfacez=rfacey+blockDim.x;
  float *rfomegaacex=rfacez+blockDim.x;
  float *rfomegaacey=rfomegaacex+blockDim.x;
  float *rfomegaacez=rfomegaacey+blockDim.x;

  const unsigned tid=threadIdx.x;                      //-Thread number.
  const unsigned cf=blockIdx.y*gridDim.x + blockIdx.x; //-Floating number.
  
  //-Loads floating data.
  float4 rfdata=ftodata[cf];
  const unsigned fpini=(unsigned)__float_as_int(rfdata.x);
  const unsigned fnp=(unsigned)__float_as_int(rfdata.y);
  const float fradius=rfdata.z;
  const double3 rcenter=ftocenter[cf];

  //-Initialises shared memory to zero.
  const unsigned ntid=(fnp<blockDim.x? fnp: blockDim.x); //-Number of used threads. | Numero de threads utilizados.
  if(tid<ntid){
    rfacex[tid]=rfacey[tid]=rfacez[tid]=0;
    rfomegaacex[tid]=rfomegaacey[tid]=rfomegaacez[tid]=0;
  }

  //-Computes data in shared memory. | Calcula datos en memoria shared.
  const unsigned nfor=unsigned((fnp+blockDim.x-1)/blockDim.x);
  for(unsigned cfor=0;cfor<nfor;cfor++){
    unsigned p=cfor*blockDim.x+tid;
    if(p<fnp){
      const unsigned rp=ftridp[p+fpini];
      if(rp!=UINT_MAX){
        float3 race=ace[rp];
        race.x-=gravity.x; race.y-=gravity.y; race.z-=gravity.z;
        rfacex[tid]+=race.x; rfacey[tid]+=race.y; rfacez[tid]+=race.z;
        //-Computes distance from the centre. | Calcula distancia al centro.
        double2 rposxy=posxy[rp];
        float dx,dy,dz;
        KerFtPeriodicDist<periactive>(rposxy.x,rposxy.y,posz[rp],rcenter.x,rcenter.y,rcenter.z,fradius,dx,dy,dz);
        //-Computes omegaace.
        rfomegaacex[tid]+=(race.z*dy - race.y*dz);
        rfomegaacey[tid]+=(race.x*dz - race.z*dx);
        rfomegaacez[tid]+=(race.y*dx - race.x*dy);
      }
    }
  }

  //-Reduces data in shared memory and stores results.
  //-Reduce datos de memoria shared y guarda resultados.
  __syncthreads();
  if(!tid){
    float3 face=make_float3(0,0,0);
    float3 fomegaace=make_float3(0,0,0);
    for(unsigned c=0;c<ntid;c++){
      face.x+=rfacex[c];  face.y+=rfacey[c];  face.z+=rfacez[c];
      fomegaace.x+=rfomegaacex[c]; fomegaace.y+=rfomegaacey[c]; fomegaace.z+=rfomegaacez[c];
    }
    //-Stores results in ftoforcessum[].
    ftoforcessum[cf*2]=face;
    ftoforcessum[cf*2+1]=fomegaace;
  }
}

//==============================================================================
/// Calculate summation: face, fomegaace in ftoforcessum[].
/// Calcula suma de face y fomegaace a partir de particulas floating en ftoforcessum[].
//==============================================================================
void FtCalcForcesSum(bool periactive,unsigned ftcount
  ,tfloat3 gravity,const float4 *ftodata,const double3 *ftocenter,const unsigned *ftridp
  ,const double2 *posxy,const double *posz,const float3 *ace
  ,float3 *ftoforcessum)
{
  if(ftcount){
    const unsigned bsize=256;
    const unsigned smem=sizeof(float)*(3+3)*bsize;
    dim3 sgrid=GetGridSize(ftcount*bsize,bsize);
    if(periactive)KerFtCalcForcesSum<true>  <<<sgrid,bsize,smem>>> (Float3(gravity),ftodata,ftocenter,ftridp,posxy,posz,ace,ftoforcessum);
    else          KerFtCalcForcesSum<false> <<<sgrid,bsize,smem>>> (Float3(gravity),ftodata,ftocenter,ftridp,posxy,posz,ace,ftoforcessum);
  }
}

//------------------------------------------------------------------------------
/// Carga valores de matriz 3x3 en bloques de 4, 4 y 1.
/// Loads values of matrix 3x3 in blocks of 4, 4 y 1.
//------------------------------------------------------------------------------
__device__ void KerLoadMatrix3f(unsigned c,const float4 *data8,const float *data1,tmatrix3f &v)
{
  float4 v4=data8[c*2];
  v.a11=v4.x; v.a12=v4.y; v.a13=v4.z; v.a21=v4.w;
  v4=data8[c*2+1];
  v.a22=v4.x; v.a23=v4.y; v.a31=v4.z; v.a32=v4.w;
  v.a33=data1[c];
}

//------------------------------------------------------------------------------
/// Calculate forces around floating object particles.
/// Calcula fuerzas sobre floatings.
//------------------------------------------------------------------------------
__global__ void KerFtCalcForces(unsigned ftcount,float3 gravity,const float4 *ftodata
  ,const float3 *ftoangles,const float4 *ftoinertiaini8,const float *ftoinertiaini1
  ,const float3 *ftoforcessum,float3 *ftoforces) //fdata={pini,np,radius,mass}
{
  unsigned cf=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of floating.
  if(cf<ftcount){
    //-Loads floating data.
    const float fmass=ftodata[cf].w;
    const float3 fang=ftoangles[cf];
    tmatrix3f inert;
    KerLoadMatrix3f(cf,ftoinertiaini8,ftoinertiaini1,inert);

    //-Compute a cumulative rotation matrix.
    const tmatrix3f frot=cumath::RotMatrix3x3(fang);
    //-Compute the intertia tensor by rotating the initial tensor to the curent orientation I=(R*I_0)*R^T.
    inert=cumath::MulMatrix3x3(cumath::MulMatrix3x3(frot,inert),cumath::TrasMatrix3x3(frot));
    //-Calculates the inverse of the intertia matrix to compute the I^-1 * L= W
    const tmatrix3f invinert=cumath::InverseMatrix3x3(inert);

    //-Loads traslational and rotational velocities.
    float3 face=ftoforcessum[cf*2];
    float3 fomegaace=ftoforcessum[cf*2+1];

    //-Calculate omega starting from fomegaace & invinert. | Calcula omega a partir de fomegaace y invinert.
    {
      float3 omegaace;
      omegaace.x=(fomegaace.x*invinert.a11+fomegaace.y*invinert.a12+fomegaace.z*invinert.a13);
      omegaace.y=(fomegaace.x*invinert.a21+fomegaace.y*invinert.a22+fomegaace.z*invinert.a23);
      omegaace.z=(fomegaace.x*invinert.a31+fomegaace.y*invinert.a32+fomegaace.z*invinert.a33);
      fomegaace=omegaace;
    }
    //-Add gravity and divide by mass. | Añade gravedad y divide por la masa.
    face.x=(face.x+fmass*gravity.x)/fmass;
    face.y=(face.y+fmass*gravity.y)/fmass;
    face.z=(face.z+fmass*gravity.z)/fmass;
    //-Stores results in ftoforces[].
    float3 *ftoforcesc=(ftoforces+(cf*2));
    const float3 rface=ftoforcesc[0];
    const float3 rfome=ftoforcesc[1];
    face.x+=rface.x;      face.y+=rface.y;      face.z+=rface.z;
    fomegaace.x+=rfome.x; fomegaace.y+=rfome.y; fomegaace.z+=rfome.z;
    ftoforcesc[0]=face;
    ftoforcesc[1]=fomegaace;
  }
}

//==============================================================================
/// Computes forces on floatings.
/// Calcula fuerzas sobre floatings.
//==============================================================================
void FtCalcForces(unsigned ftcount,tfloat3 gravity,const float4 *ftodata
  ,const float3 *ftoangles,const float4 *ftoinertiaini8,const float *ftoinertiaini1
  ,const float3 *ftoforcessum,float3 *ftoforces)
{
  if(ftcount){
    dim3 sgrid=GetGridSize(ftcount,SPHBSIZE);
    KerFtCalcForces <<<sgrid,SPHBSIZE>>> (ftcount,Float3(gravity),ftodata,ftoangles,ftoinertiaini8,ftoinertiaini1,ftoforcessum,ftoforces);
  }
}


//------------------------------------------------------------------------------
/// Calculate data to update floatings.
/// Calcula datos para actualizar floatings.
//------------------------------------------------------------------------------
__global__ void KerFtCalcForcesRes(unsigned ftcount,bool simulate2d,double dt
  ,const float3 *ftoomega,const float3 *ftovel,const double3 *ftocenter,const float3 *ftoforces
  ,float3 *ftoforcesres,double3 *ftocenterres)
{
  const unsigned cf=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Floating number.
  if(cf<ftcount){
    //-Compute fomega.
    float3 fomega=ftoomega[cf];
    {
      const float3 omegaace=ftoforces[cf*2+1];
      fomega.x=float(dt*omegaace.x+fomega.x);
      fomega.y=float(dt*omegaace.y+fomega.y);
      fomega.z=float(dt*omegaace.z+fomega.z);
    }
    float3 fvel=ftovel[cf];
    //-Zero components for 2-D simulation. | Anula componentes para 2D.
    float3 face=ftoforces[cf*2];
    if(simulate2d){ face.y=0; fomega.x=0; fomega.z=0; fvel.y=0; }
    //-Compute fcenter.
    double3 fcenter=ftocenter[cf];
    fcenter.x+=dt*fvel.x;
    fcenter.y+=dt*fvel.y;
    fcenter.z+=dt*fvel.z;
    //-Compute fvel.
    fvel.x=float(dt*face.x+fvel.x);
    fvel.y=float(dt*face.y+fvel.y);
    fvel.z=float(dt*face.z+fvel.z);
    //-Store data to update floating. | Guarda datos para actualizar floatings.
    ftoforcesres[cf*2]=fomega;
    ftoforcesres[cf*2+1]=fvel;
    ftocenterres[cf]=fcenter;
  }
}

//==============================================================================
/// Computes forces on floatings.
/// Calcula fuerzas sobre floatings.
//==============================================================================
void FtCalcForcesRes(unsigned ftcount,bool simulate2d,double dt
  ,const float3 *ftoomega,const float3 *ftovel,const double3 *ftocenter,const float3 *ftoforces
  ,float3 *ftoforcesres,double3 *ftocenterres)
{
  if(ftcount){
    dim3 sgrid=GetGridSize(ftcount,SPHBSIZE);
    KerFtCalcForcesRes <<<sgrid,SPHBSIZE>>> (ftcount,simulate2d,dt,ftoomega,ftovel,ftocenter,ftoforces,ftoforcesres,ftocenterres);
  }
}

//------------------------------------------------------------------------------
/// Updates information and particles of floating bodies.
//------------------------------------------------------------------------------
template<bool periactive> __global__ void KerFtUpdate(bool predictor,double dt //fdata={pini,np,radius,mass}
  ,const float4 *ftodata,const float3 *ftoforcesres,double3 *ftocenterres,const unsigned *ftridp
  ,double3 *ftocenter,float3 *ftoangles,float3 *ftovel,float3 *ftoomega
  ,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  const unsigned tid=threadIdx.x;                      //-Thread number.
  const unsigned cf=blockIdx.y*gridDim.x + blockIdx.x; //-Floating number.
  //-Obtains floating data.
  const float3 fomega=ftoforcesres[cf*2];
  const float3 fvel=ftoforcesres[cf*2+1];
  const double3 fcenter=ftocenterres[cf];
  float4 rfdata=ftodata[cf];
  const unsigned fpini=(unsigned)__float_as_int(rfdata.x);
  const unsigned fnp=(unsigned)__float_as_int(rfdata.y);
  const float fradius=rfdata.z;
  //-Updates floating particles.
  const unsigned nfor=unsigned((fnp+blockDim.x-1)/blockDim.x);
  for(unsigned cfor=0;cfor<nfor;cfor++){
    unsigned fp=cfor*blockDim.x+tid;
    if(fp<fnp){
      const unsigned p=ftridp[fp+fpini];
      if(p!=UINT_MAX){
        double2 rposxy=posxy[p];
        double rposz=posz[p];
        float4 rvel=velrhop[p];
        //-Computes and stores position displacement.
        const double dx=dt*double(rvel.x);
        const double dy=dt*double(rvel.y);
        const double dz=dt*double(rvel.z);
        KerUpdatePos<periactive>(rposxy,rposz,dx,dy,dz,false,p,posxy,posz,dcell,code);
        //-Computes and stores new velocity.
        float disx,disy,disz;
        KerFtPeriodicDist<periactive>(rposxy.x+dx,rposxy.y+dy,rposz+dz,fcenter.x,fcenter.y,fcenter.z,fradius,disx,disy,disz);
        rvel.x=fvel.x+(fomega.y*disz-fomega.z*disy);
        rvel.y=fvel.y+(fomega.z*disx-fomega.x*disz);
        rvel.z=fvel.z+(fomega.x*disy-fomega.y*disx);
        velrhop[p]=rvel;
      }
    }
  }

  //-Stores floating data.
  __syncthreads();
  if(!tid && !predictor){
    ftocenter[cf]=(periactive? KerUpdatePeriodicPos(fcenter): fcenter);
    float3 rangles=ftoangles[cf];
    rangles.x=float(double(rangles.x)+double(fomega.x)*dt);
    rangles.y=float(double(rangles.y)+double(fomega.y)*dt);
    rangles.z=float(double(rangles.z)+double(fomega.z)*dt);
    ftoangles[cf]=rangles;
    ftovel[cf]=fvel;
    ftoomega[cf]=fomega;
  }
}

//==============================================================================
/// Updates information and particles of floating bodies.
//==============================================================================
void FtUpdate(bool periactive,bool predictor,unsigned ftcount,double dt
  ,const float4 *ftodata,const float3 *ftoforcesres,double3 *ftocenterres,const unsigned *ftridp
  ,double3 *ftocenter,float3 *ftoangles,float3 *ftovel,float3 *ftoomega
  ,double2 *posxy,double *posz,unsigned *dcell,float4 *velrhop,typecode *code)
{
  if(ftcount){
    const unsigned bsize=128; 
    dim3 sgrid=GetGridSize(ftcount*bsize,bsize);
    if(periactive)KerFtUpdate<true>  <<<sgrid,bsize>>> (predictor,dt,ftodata,ftoforcesres,ftocenterres,ftridp,ftocenter,ftoangles,ftovel,ftoomega,posxy,posz,dcell,velrhop,code);
    else          KerFtUpdate<false> <<<sgrid,bsize>>> (predictor,dt,ftodata,ftoforcesres,ftocenterres,ftridp,ftocenter,ftoangles,ftovel,ftoomega,posxy,posz,dcell,velrhop,code);
  }
}


//##############################################################################
//# Kernels for Periodic conditions
//# Kernels para Periodic conditions
//##############################################################################
//------------------------------------------------------------------------------
/// Marks current periodics to be ignored.
/// Marca las periodicas actuales como ignorar.
//------------------------------------------------------------------------------
__global__ void KerPeriodicIgnore(unsigned n,typecode *code)
{
  const unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    //-Checks code of particles.
    //-Comprueba codigo de particula.
    const typecode rcode=code[p];
    if(CODE_IsPeriodic(rcode))code[p]=CODE_SetOutIgnore(rcode);
  }
}

//==============================================================================
/// Marks current periodics to be ignored.
/// Marca las periodicas actuales como ignorar.
//==============================================================================
void PeriodicIgnore(unsigned n,typecode *code){
  if(n){
    dim3 sgrid=GetGridSize(n,SPHBSIZE);
    KerPeriodicIgnore <<<sgrid,SPHBSIZE>>> (n,code);
  }
}

//------------------------------------------------------------------------------
/// Create list of new periodic particles to be duplicated and 
/// marks old periodics to be ignored.
///
/// Crea lista de nuevas particulas periodicas a duplicar y con delper activado
/// marca las periodicas viejas para ignorar.
//------------------------------------------------------------------------------
__global__ void KerPeriodicMakeList(unsigned n,unsigned pini,unsigned nmax
  ,double3 mapposmin,double3 mapposmax,double3 perinc
  ,const double2 *posxy,const double *posz,const typecode *code,unsigned *listp)
{
  extern __shared__ unsigned slist[];
  if(!threadIdx.x)slist[0]=0;
  __syncthreads();
  const unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p2=p+pini;
    //-Inteacts with normal or periodic particles.
    //-Se queda con particulas normales o periodicas.
    if(CODE_GetSpecialValue(code[p2])<=CODE_PERIODIC){
      //-Obtains particle position.
      const double2 rxy=posxy[p2];
      const double rx=rxy.x,ry=rxy.y;
      const double rz=posz[p2];
      double rx2=rx+perinc.x,ry2=ry+perinc.y,rz2=rz+perinc.z;
      if(mapposmin.x<=rx2 && mapposmin.y<=ry2 && mapposmin.z<=rz2 && rx2<mapposmax.x && ry2<mapposmax.y && rz2<mapposmax.z){
        unsigned cp=atomicAdd(slist,1);  slist[cp+1]=p2;
      }
      rx2=rx-perinc.x; ry2=ry-perinc.y; rz2=rz-perinc.z;
      if(mapposmin.x<=rx2 && mapposmin.y<=ry2 && mapposmin.z<=rz2 && rx2<mapposmax.x && ry2<mapposmax.y && rz2<mapposmax.z){
        unsigned cp=atomicAdd(slist,1);  slist[cp+1]=(p2|0x80000000);
      }
    }
  }
  __syncthreads();
  const unsigned ns=slist[0];
  __syncthreads();
  if(!threadIdx.x && ns)slist[0]=atomicAdd((listp+nmax),ns);
  __syncthreads();
  if(threadIdx.x<ns){
    unsigned cp=slist[0]+threadIdx.x;
    if(cp<nmax)listp[cp]=slist[threadIdx.x+1];
  }
  if(blockDim.x+threadIdx.x<ns){ //-There may be twice as many periodics per thread. | Puede haber el doble de periodicas que threads.
    unsigned cp=blockDim.x+slist[0]+threadIdx.x;
    if(cp<nmax)listp[cp]=slist[blockDim.x+threadIdx.x+1];
  }
}

//==============================================================================
/// Create list of new periodic particles to be duplicated.
/// With stable activated reorders perioc list.
///
/// Crea lista de nuevas particulas periodicas a duplicar.
/// Con stable activado reordena lista de periodicas.
//==============================================================================
unsigned PeriodicMakeList(unsigned n,unsigned pini,bool stable,unsigned nmax
  ,tdouble3 mapposmin,tdouble3 mapposmax,tdouble3 perinc
  ,const double2 *posxy,const double *posz,const typecode *code,unsigned *listp)
{
  unsigned count=0;
  if(n){
    //-lspg size list initialized to zero.
    //-Inicializa tamaño de lista lspg a cero.
    hipMemset(listp+nmax,0,sizeof(unsigned));
    dim3 sgrid=GetGridSize(n,SPHBSIZE);
    const unsigned smem=(SPHBSIZE*2+1)*sizeof(unsigned); //-Each particle can leave two new periodic over the counter position. | De cada particula pueden salir 2 nuevas periodicas mas la posicion del contador.
    KerPeriodicMakeList <<<sgrid,SPHBSIZE,smem>>> (n,pini,nmax,Double3(mapposmin),Double3(mapposmax),Double3(perinc),posxy,posz,code,listp);
    hipMemcpy(&count,listp+nmax,sizeof(unsigned),hipMemcpyDeviceToHost);
    //-Reorders list if it is valid and stable has been activated.
    //-Reordena lista si es valida y stable esta activado.
    if(stable && count && count<=nmax){
      thrust::device_ptr<unsigned> dev_list(listp);
      thrust::sort(dev_list,dev_list+count);
    }
  }
  return(count);
}

//------------------------------------------------------------------------------
/// Doubles the position of the indicated particle using a displacement.
/// Duplicate particles are considered valid and are always within
/// the domain.
/// This kernel applies to single-GPU and multi-GPU because the calculations are made
/// from domposmin.
/// It controls the cell coordinates not exceed the maximum.
///
/// Duplica la posicion de la particula indicada aplicandole un desplazamiento.
/// Las particulas duplicadas se considera que siempre son validas y estan dentro
/// del dominio.
/// Este kernel vale para single-gpu y multi-gpu porque los calculos se hacen 
/// a partir de domposmin.
/// Se controla que las coordendas de celda no sobrepasen el maximo.
//------------------------------------------------------------------------------
__device__ void KerPeriodicDuplicatePos(unsigned pnew,unsigned pcopy
  ,bool inverse,double dx,double dy,double dz,uint3 cellmax
  ,double2 *posxy,double *posz,unsigned *dcell)
{
  //-Obtains position of the particle to be duplicated.
  //-Obtiene pos de particula a duplicar.
  double2 rxy=posxy[pcopy];
  double rz=posz[pcopy];
  //-Applies displacement.
  rxy.x+=(inverse? -dx: dx);
  rxy.y+=(inverse? -dy: dy);
  rz+=(inverse? -dz: dz);
  //-Computes cell coordinates within the domain.
  //-Calcula coordendas de celda dentro de dominio.
  unsigned cx=unsigned((rxy.x-CTE.domposminx)/CTE.scell);
  unsigned cy=unsigned((rxy.y-CTE.domposminy)/CTE.scell);
  unsigned cz=unsigned((rz-CTE.domposminz)/CTE.scell);
  //-Adjust cell coordinates if they exceed the maximum.
  //-Ajusta las coordendas de celda si sobrepasan el maximo.
  cx=(cx<=cellmax.x? cx: cellmax.x);
  cy=(cy<=cellmax.y? cy: cellmax.y);
  cz=(cz<=cellmax.z? cz: cellmax.z);
  //-Stores position and cell of the new particles.
  //-Graba posicion y celda de nuevas particulas.
  posxy[pnew]=rxy;
  posz[pnew]=rz;
  dcell[pnew]=PC__Cell(CTE.cellcode,cx,cy,cz);
}

//------------------------------------------------------------------------------
/// Creates periodic particles from a list of particles to duplicate.
/// It is assumed that all particles are valid.
/// This kernel applies to single-GPU and multi-GPU because it uses domposmin.
///
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
/// Se presupone que todas las particulas son validas.
/// Este kernel vale para single-gpu y multi-gpu porque usa domposmin. 
//------------------------------------------------------------------------------
__global__ void KerPeriodicDuplicateVerlet(unsigned n,unsigned pini,uint3 cellmax,double3 perinc
  ,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,tsymatrix3f *spstau,float4 *velrhopm1)
{
  const unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned pnew=p+pini;
    const unsigned rp=listp[p];
    const unsigned pcopy=(rp&0x7FFFFFFF);
    //-Adjusts cell position of the new particles.
    //-Ajusta posicion y celda de nueva particula.
    KerPeriodicDuplicatePos(pnew,pcopy,(rp>=0x80000000),perinc.x,perinc.y,perinc.z,cellmax,posxy,posz,dcell);
    //-Copies the remaining data.
    //-Copia el resto de datos.
    idp[pnew]=idp[pcopy];
    code[pnew]=CODE_SetPeriodic(code[pcopy]);
    velrhop[pnew]=velrhop[pcopy];
    velrhopm1[pnew]=velrhopm1[pcopy];
    if(spstau)spstau[pnew]=spstau[pcopy];
  }
}

//==============================================================================
/// Creates periodic particles from a list of particles to duplicate.
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
//==============================================================================
void PeriodicDuplicateVerlet(unsigned n,unsigned pini,tuint3 domcells,tdouble3 perinc
  ,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,tsymatrix3f *spstau,float4 *velrhopm1)
{
  if(n){
    uint3 cellmax=make_uint3(domcells.x-1,domcells.y-1,domcells.z-1);
    dim3 sgrid=GetGridSize(n,SPHBSIZE);
    KerPeriodicDuplicateVerlet <<<sgrid,SPHBSIZE>>> (n,pini,cellmax,Double3(perinc),listp,idp,code,dcell,posxy,posz,velrhop,spstau,velrhopm1);
  }
}

//------------------------------------------------------------------------------
/// Creates periodic particles from a list of particles to duplicate.
/// It is assumed that all particles are valid.
/// This kernel applies to single-GPU and multi-GPU because it uses domposmin.
///
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
/// Se presupone que todas las particulas son validas.
/// Este kernel vale para single-gpu y multi-gpu porque usa domposmin. 
//------------------------------------------------------------------------------
template<bool varspre> __global__ void KerPeriodicDuplicateSymplectic(unsigned n,unsigned pini
  ,uint3 cellmax,double3 perinc,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,tsymatrix3f *spstau,double2 *posxypre,double *poszpre,float4 *velrhoppre)
{
  const unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned pnew=p+pini;
    const unsigned rp=listp[p];
    const unsigned pcopy=(rp&0x7FFFFFFF);
    //-Adjusts cell position of the new particles.
    //-Ajusta posicion y celda de nueva particula.
    KerPeriodicDuplicatePos(pnew,pcopy,(rp>=0x80000000),perinc.x,perinc.y,perinc.z,cellmax,posxy,posz,dcell);
    //-Copies the remaining data.
    //-Copia el resto de datos.
    idp[pnew]=idp[pcopy];
    code[pnew]=CODE_SetPeriodic(code[pcopy]);
    velrhop[pnew]=velrhop[pcopy];
    if(varspre){
      posxypre[pnew]=posxypre[pcopy];
      poszpre[pnew]=poszpre[pcopy];
      velrhoppre[pnew]=velrhoppre[pcopy];
    }
    if(spstau)spstau[pnew]=spstau[pcopy];
  }
}

//==============================================================================
/// Creates periodic particles from a list of particles to duplicate.
/// Crea particulas periodicas a partir de una lista con las particulas a duplicar.
//==============================================================================
void PeriodicDuplicateSymplectic(unsigned n,unsigned pini
  ,tuint3 domcells,tdouble3 perinc,const unsigned *listp,unsigned *idp,typecode *code,unsigned *dcell
  ,double2 *posxy,double *posz,float4 *velrhop,tsymatrix3f *spstau,double2 *posxypre,double *poszpre,float4 *velrhoppre)
{
  if(n){
    uint3 cellmax=make_uint3(domcells.x-1,domcells.y-1,domcells.z-1);
    dim3 sgrid=GetGridSize(n,SPHBSIZE);
    if(posxypre!=NULL)KerPeriodicDuplicateSymplectic<true>  <<<sgrid,SPHBSIZE>>> (n,pini,cellmax,Double3(perinc),listp,idp,code,dcell,posxy,posz,velrhop,spstau,posxypre,poszpre,velrhoppre);
    else              KerPeriodicDuplicateSymplectic<false> <<<sgrid,SPHBSIZE>>> (n,pini,cellmax,Double3(perinc),listp,idp,code,dcell,posxy,posz,velrhop,spstau,posxypre,poszpre,velrhoppre);
  }
}


//##############################################################################
//# Kernels for external forces (JSphAccInput).
//# Kernels para external forces (JSphAccInput).
//##############################################################################
//------------------------------------------------------
/// Adds variable forces to particle sets.
//------------------------------------------------------
__global__ void KerAddAccInputAng(unsigned n,unsigned pini,typecode codesel,float3 gravity
  ,bool setgravity,double3 acclin,double3 accang,double3 centre,double3 velang,double3 vellin
  ,const typecode *code,const double2 *posxy,const double *posz,const float4 *velrhop,float3 *ace)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
  if(p<n){
    p+=pini;
    //Check if the current particle is part of the particle set by its Mk.
    if(CODE_GetTypeValue(code[p])==codesel){
      const float3 accf=ace[p]; //-Gets the current particles acceleration value.
      double accx=accf.x,accy=accf.y,accz=accf.z;
      //-Adds linear acceleration.
      accx+=acclin.x;  accy+=acclin.y;  accz+=acclin.z;
      //-Subtract global gravity from the acceleration if it is set in the input file.
      if(!setgravity){
        accx-=gravity.x;  accy-=gravity.y;  accz-=gravity.z; 
      }

      //-Adds angular acceleration.
      const double2 rxy=posxy[p];
      const double dcx=rxy.x-centre.x;
      const double dcy=rxy.y-centre.y;
      const double dcz=posz[p]-centre.z;
      //-Get the current particle's velocity.
      const float4 rvel=velrhop[p];
      const double velx=rvel.x;
      const double vely=rvel.y;
      const double velz=rvel.z;

      //-Calculate angular acceleration ((Dw/Dt) x (r_i - r)) + (w x (w x (r_i - r))) + (2w x (v_i - v))
      //(Dw/Dt) x (r_i - r) (term1)
      accx+=(accang.y*dcz)-(accang.z*dcy);
      accy+=(accang.z*dcx)-(accang.x*dcz);
      accz+=(accang.x*dcy)-(accang.y*dcx);

      //-Centripetal acceleration (term2).
      //-First find w x (r_i - r)).
      const double innerx=(velang.y*dcz)-(velang.z*dcy);
      const double innery=(velang.z*dcx)-(velang.x*dcz);
      const double innerz=(velang.x*dcy)-(velang.y*dcx);
      //-Find w x inner.
      accx+=(velang.y*innerz)-(velang.z*innery);
      accy+=(velang.z*innerx)-(velang.x*innerz);
      accz+=(velang.x*innery)-(velang.y*innerx);

      //-Coriolis acceleration 2w x (v_i - v) (term3).
      accx+=((2.0*velang.y)*velz)-((2.0*velang.z)*(vely-vellin.y));
      accy+=((2.0*velang.z)*velx)-((2.0*velang.x)*(velz-vellin.z));
      accz+=((2.0*velang.x)*vely)-((2.0*velang.y)*(velx-vellin.x));

      //-Stores the new acceleration value.
      ace[p]=make_float3(float(accx),float(accy),float(accz));
    }
  }
}

//------------------------------------------------------
/// Adds variable forces to particle sets.
//------------------------------------------------------
__global__ void KerAddAccInputLin(unsigned n,unsigned pini,typecode codesel,float3 gravity
  ,bool setgravity,double3 acclin,const typecode *code,float3 *ace)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
  if(p<n){
    p+=pini;
    //-Check if the current particle is part of the particle set by its Mk.
    if(CODE_GetTypeValue(code[p])==codesel){
      const float3 accf=ace[p]; //-Gets the current particles acceleration value.
      double accx=accf.x,accy=accf.y,accz=accf.z;
      //-Adds linear acceleration.
      accx+=acclin.x;  accy+=acclin.y;  accz+=acclin.z;
      //-Subtract global gravity from the acceleration if it is set in the input file.
      if(!setgravity){
        accx-=gravity.x;  accy-=gravity.y;  accz-=gravity.z; 
      }
      //-Stores the new acceleration value.
      ace[p]=make_float3(float(accx),float(accy),float(accz));
    }
  }
}

//==================================================================================================
/// Adds variable acceleration forces for particle MK groups that have an input file.
//==================================================================================================
void AddAccInput(unsigned n,unsigned pini,typecode codesel
  ,tdouble3 acclin,tdouble3 accang,tdouble3 centre,tdouble3 velang,tdouble3 vellin,bool setgravity
  ,tfloat3 gravity,const typecode *code,const double2 *posxy,const double *posz,const float4 *velrhop,float3 *ace)
{
  if(n){
    dim3 sgrid=GetGridSize(n,SPHBSIZE);
    const bool withaccang=(accang.x!=0 || accang.y!=0 || accang.z!=0);
    if(withaccang)KerAddAccInputAng <<<sgrid,SPHBSIZE>>> (n,pini,codesel,Float3(gravity),setgravity,Double3(acclin),Double3(accang),Double3(centre),Double3(velang),Double3(vellin),code,posxy,posz,velrhop,ace);
    else          KerAddAccInputLin <<<sgrid,SPHBSIZE>>> (n,pini,codesel,Float3(gravity),setgravity,Double3(acclin),code,ace);
  }
}


//##############################################################################
//# Kernels for Damping.
//##############################################################################
//------------------------------------------------------------------------------
/// Returns TRUE when code==NULL or particle is normal and fluid.
//------------------------------------------------------------------------------
__device__ bool KerIsNormalFluid(const typecode *code,unsigned p){
  if(code){//-Descarta particulas floating o periodicas.
    const typecode cod=code[p];
    return(CODE_IsNormal(cod) && CODE_IsFluid(cod));
  }
  return(true);
}
//------------------------------------------------------------------------------
/// Solves point on the plane.
/// Resuelve punto en el plano.
//------------------------------------------------------------------------------
__device__ double KerPointPlane(const double4 &pla,double px,double py,double pz)
{
  return(pla.x*px+pla.y*py+pla.z*pz+pla.w);
}
//------------------------------------------------------------------------------
/// Solves point on the plane.
/// Resuelve punto en el plano.
//------------------------------------------------------------------------------
__device__ double KerPointPlane(const double4 &pla,const double3 &pt)
{
  return(pla.x*pt.x+pla.y*pt.y+pla.z*pt.z+pla.w);
}

//------------------------------------------------------------------------------
/// Applies Damping.
/// Aplica Damping.
//------------------------------------------------------------------------------
__global__ void KerComputeDamping(unsigned n,unsigned pini
  ,double dt,double4 plane,float dist,float over,float3 factorxyz,float redumax
  ,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p1=p+pini;
    const bool ok=KerIsNormalFluid(code,p1);//-Ignore floating and periodic particles. | Descarta particulas floating o periodicas.
    if(ok){
      const double2 rposxy=posxy[p1];
      const double rposz=posz[p1];
      double vdis=KerPointPlane(plane,rposxy.x,rposxy.y,rposz);  //fmath::PointPlane(plane,ps);
      if(0<vdis && vdis<=dist+over){
        const double fdis=(vdis>=dist? 1.: vdis/dist);
        const double redudt=dt*(fdis*fdis)*redumax;
        double redudtx=(1.-redudt*factorxyz.x);
        double redudty=(1.-redudt*factorxyz.y);
        double redudtz=(1.-redudt*factorxyz.z);
        redudtx=(redudtx<0? 0.: redudtx);
        redudty=(redudty<0? 0.: redudty);
        redudtz=(redudtz<0? 0.: redudtz);
        float4 rvel=velrhop[p1];
        rvel.x=float(redudtx*rvel.x); 
        rvel.y=float(redudty*rvel.y); 
        rvel.z=float(redudtz*rvel.z); 
        velrhop[p1]=rvel;
      }
    }
  }
}
//==============================================================================
/// Applies Damping.
/// Aplica Damping.
//==============================================================================
void ComputeDamping(double dt,tdouble4 plane,float dist,float over,tfloat3 factorxyz,float redumax
  ,unsigned n,unsigned pini,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  if(n){
    dim3 sgridf=GetGridSize(n,SPHBSIZE);
    KerComputeDamping <<<sgridf,SPHBSIZE>>> (n,pini,dt,Double4(plane),dist,over,Float3(factorxyz),redumax
      ,posxy,posz,code,velrhop);
  }
}

//------------------------------------------------------------------------------
/// Applies Damping to limited domain.
/// Aplica Damping limitado a un dominio.
//------------------------------------------------------------------------------
__global__ void KerComputeDampingPla(unsigned n,unsigned pini
  ,double dt,double4 plane,float dist,float over,float3 factorxyz,float redumax
  ,double zmin,double zmax,double4 pla0,double4 pla1,double4 pla2,double4 pla3
  ,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  unsigned p=blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    const unsigned p1=p+pini;
    const bool ok=KerIsNormalFluid(code,p1);//-Ignore floating and periodic particles. | Descarta particulas floating o periodicas.
    if(ok){
      const double2 rposxy=posxy[p1];
      const double rposz=posz[p1];
      const double3 ps=make_double3(rposxy.x,rposxy.y,rposz);
      double vdis=KerPointPlane(plane,ps);  //fmath::PointPlane(plane,ps);
      if(0<vdis && vdis<=dist+over){
        if(ps.z>=zmin && ps.z<=zmax && KerPointPlane(pla0,ps)<=0 && KerPointPlane(pla1,ps)<=0 && KerPointPlane(pla2,ps)<=0 && KerPointPlane(pla3,ps)<=0){
          const double fdis=(vdis>=dist? 1.: vdis/dist);
          const double redudt=dt*(fdis*fdis)*redumax;
          double redudtx=(1.-redudt*factorxyz.x);
          double redudty=(1.-redudt*factorxyz.y);
          double redudtz=(1.-redudt*factorxyz.z);
          redudtx=(redudtx<0? 0.: redudtx);
          redudty=(redudty<0? 0.: redudty);
          redudtz=(redudtz<0? 0.: redudtz);
          float4 rvel=velrhop[p1];
          rvel.x=float(redudtx*rvel.x); 
          rvel.y=float(redudty*rvel.y); 
          rvel.z=float(redudtz*rvel.z); 
          velrhop[p1]=rvel;
        }
      }
    }
  }
}
//==============================================================================
/// Applies Damping to limited domain.
/// Aplica Damping limitado a un dominio.
//==============================================================================
void ComputeDampingPla(double dt,tdouble4 plane,float dist,float over,tfloat3 factorxyz,float redumax
  ,double zmin,double zmax,tdouble4 pla0,tdouble4 pla1,tdouble4 pla2,tdouble4 pla3
  ,unsigned n,unsigned pini,const double2 *posxy,const double *posz,const typecode *code
  ,float4 *velrhop)
{
  if(n){
    dim3 sgridf=GetGridSize(n,SPHBSIZE);
    KerComputeDampingPla <<<sgridf,SPHBSIZE>>> (n,pini,dt,Double4(plane),dist,over,Float3(factorxyz),redumax
      ,zmin,zmax,Double4(pla0),Double4(pla1),Double4(pla2),Double4(pla3)
      ,posxy,posz,code,velrhop);
  }
}


}


